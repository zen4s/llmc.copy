
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#undef NDEBUG
#include "assert.h"
#include "float.h"
#include "stdio.h"

struct SplitFloatResult {
    hip_bfloat16 b_float;
    unsigned short bits;
};

template<class T, class S>
__host__ __device__ T bit_cast(S v) {
    T dest;
    static_assert(sizeof(v) == sizeof(dest));
    memcpy(&dest, &v, sizeof(v));
    return dest;
}

__host__  __device__ unsigned int float_as_uint(float f) {
    return bit_cast<unsigned int>(f);
}

__host__  __device__ unsigned short bfloat16_as_ushort(hip_bfloat16 f) {
    return bit_cast<unsigned short>(f);
}

__host__  __device__ float uint_as_float(unsigned int u) {
    return bit_cast<float>(u);
}

__host__  __device__ hip_bfloat16 ushort_as_bfloat16(unsigned short u) {
    return bit_cast<hip_bfloat16>(u);
}

// Splits a float into a bfloat16 and the remaining significant bits
__host__  __device__ SplitFloatResult split_float(float value, unsigned short threshold) {
    unsigned int float_bits = float_as_uint(value);
    // IEEE 754: float: S E(8) M (23)    bfloat: same, but significant 23-16 = 7 bits
    // ideally, we'd just store the cut-off 16 bits, but that doesn't work if rounding
    // is involved.
    unsigned int rounded_bits = float_bits & 0x0000FFFFu;
    if(rounded_bits > threshold) {
        SplitFloatResult result;
        result.b_float = __float2bfloat16_rn(uint_as_float(float_bits | 0xFFFFu));
        result.bits = rounded_bits & (~1u) | 1u;
        return result;
    } else {
        // truncation is easy
        SplitFloatResult result;
        result.b_float = ushort_as_bfloat16(float_bits >> 16u);
        result.bits = rounded_bits & (~1u);
        return result;
    }
}

// Reassembles a float from the bfloat16 part and the missing mantissa
__host__ __device__ float assemble_float(SplitFloatResult split) {
    constexpr const unsigned short BF16_SIGN_MASK        = 0b1'00000000'0000000u;
    constexpr const unsigned short BF16_EXPONENT_MASK    = 0b0'11111111'0000000u;
    constexpr const unsigned short BF16_SIGNIFICANT_MASK = 0b0'00000000'1111111u;
    unsigned short bf = bfloat16_as_ushort(split.b_float);
    if(split.bits & 1u) {
        // if we rounded away from zero, we need to undo these changes.
        // first, check if the significant (7 bits) of bf16 is zero
        if((bf & BF16_SIGNIFICANT_MASK) == 0) {
            // significant overflowed, need to decrement the exponent
            unsigned short exponent = (bf & BF16_EXPONENT_MASK) >> 7u;
            if(exponent == 0) {
                // zero, cannot be reached if we round away from zero
                __builtin_unreachable();
            }
            // decrement the exponent and set significant to all-ones
            bf = bf & BF16_SIGN_MASK | ((exponent-1) << 7u) | BF16_SIGNIFICANT_MASK;
        } else {
            // significant was incremented, decrement
            unsigned short significant = bf & BF16_SIGNIFICANT_MASK;
            bf = bf & (BF16_SIGN_MASK | BF16_EXPONENT_MASK) | (significant - 1);
        }
    }
    unsigned int result = (split.bits & (unsigned short)(~1u)) | (bf << 16u);
    return uint_as_float(result);
}


float round_trip(float f, unsigned short threshold) {
    SplitFloatResult split = split_float(f, threshold);
    float r = assemble_float(split);
    return r;
}

bool match_floats(float f1, float f2) {
    unsigned int u1 = float_as_uint(f1);
    unsigned int u2 = float_as_uint(f2);
    if((u1 & (~1u)) != (u2 & (~1u))) {
        printf("MISMATCH: %0b %0b\n", u1, u2);
        return false;
    }
    return true;
}

#define ASSERT_ROUND_TRIP(f) \
    assert(match_floats(f, round_trip(f, 0))); \
    assert(match_floats(f, round_trip(f, 0xFFFF)));  \

int main() {
    ASSERT_ROUND_TRIP(1.4623f)
    ASSERT_ROUND_TRIP(-63623.9f)
    ASSERT_ROUND_TRIP(FLT_TRUE_MIN)
    ASSERT_ROUND_TRIP(NAN)
    ASSERT_ROUND_TRIP(0)
    ASSERT_ROUND_TRIP(INFINITY)
    // make sure we trigger the "rounding increases exponent" code path
    float increment_exponent = bit_cast<float>((unsigned int)(0x40ff'fff0));
    ASSERT_ROUND_TRIP(increment_exponent)
    return EXIT_SUCCESS;
}