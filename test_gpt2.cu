#include "hip/hip_runtime.h"
#define TESTING
#include "platform_utils.h"
#include "train_gpt2.cu"

// poor man's tensor checker
int check_tensor(float *a, float *b, int n, char* label) {
    int print_upto = 5;
    int ok = 1;
    printf("%s\n", label);
    for (int i = 0; i < n; i++) {
        if (fabsf(a[i] - b[i]) <= 1e-2) {
            if (i < print_upto) { printf("OK "); }
        } else {
            if (i < print_upto) { printf("NOT OK "); }
            ok = 0;
        }
        if (i < print_upto) { printf("%f %f\n", a[i], b[i]); }
    }
    // print the final result
    if (ok) {
        printf("TENSOR OK\n");
    } else {
        printf("TENSOR NOT OK\n");
    }
    return ok;
}

int main(int argc, char *argv[]) {

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("[System]\n");
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    enable_tf32 = 0; // NOTE: disable TF32 for testing!!!
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M.bin");

    //int C = model.config.channels;
    int V = model.config.vocab_size;
    //int maxT = model.config.max_seq_len;
    //int L = model.config.num_layers;

    // load additional information that we will use for debugging and error checking
    FILE *state_file = fopen("gpt2_124M_debug_state.bin", "rb");
    if (state_file == NULL) { printf("Error opening state file\n"); exit(1); }
    int state_header[256];
    fread(state_header, sizeof(int), 256, state_file);
    if (state_header[0] != 20240327) { printf("Bad magic state file"); exit(1); }
    if (state_header[1] != 1) { printf("Bad version in state file"); exit(1); }
    int B = state_header[2]; // batch size, e.g. 4
    int T = state_header[3]; // time / sequence length (e.g. 64, up to maxT)
    printf("[State]\n");
    printf("batch_size: %d\n", B);
    printf("seq_len: %d\n", T);

    ParameterTensors expected_grads;
    float* expected_grads_memory = malloc_and_point_parameters(&expected_grads, model.param_sizes, 0);

    // inputs and expected outputs, only used for error checking
    int* x = (int*) malloc(B * T * sizeof(int));
    int* y = (int*) malloc(B * T * sizeof(int));
    float* expected_logits = (float*) malloc(B * T * V * sizeof(float));
    float* expected_loss = (float*) malloc(1 * sizeof(float));

    // read reference information from Python
    fread(x, sizeof(int), B*T, state_file);
    fread(y, sizeof(int), B*T, state_file);
    fread(expected_logits, sizeof(float), B*T*V, state_file);
    fread(expected_loss, sizeof(float), 1, state_file);
    fread(expected_grads_memory, sizeof(float), model.num_parameters, state_file);
    fclose(state_file);

    // overall OK signal for the test
    int allok = 1;

    // let's do 10 training iterations, following the pytorch code
    //float losses[10];
    for (int step = 0; step < 10; step++) {
        const double start_time_ms = get_time_ms();
        gpt2_forward(&model, x, y, B, T);
        const double end_time_ms = get_time_ms();
        double time_elapsed_s = (end_time_ms - start_time_ms) / 1000.0;

        if (step == 0) {
            // error checking at step 0 for reference activations

            // at this point, target should be equal to expected_logits, let's compare
            // copy logits to CPU so we can compare them
            float* logits_cpu = (float*) malloc(B * T * V * sizeof(float));
            hipMemcpy(logits_cpu, model.acts.logits, B * T * V * sizeof(float), hipMemcpyDeviceToHost);
            int logits_ok = 1;
            for (int i=0; i<B*T*V; i++) {
                if(i < 3) {
                    printf("%f %f\n", expected_logits[i], logits_cpu[i]);
                }
                if (fabsf(expected_logits[i] - logits_cpu[i]) >= 1e-2) {
                    printf("MISMATCH AT INDEX %d: ", i);
                    printf("%f %f\n", expected_logits[i],logits_cpu[i]);
                    logits_ok = 0;
                    break;
                }
            }
            if(!logits_ok) { printf("NOT "); }
            printf("OK (LOGITS)\n");
            allok = allok && logits_ok;
            free(logits_cpu);

            // compare the achieved loss
            if (fabsf(model.mean_loss - *expected_loss) >= 1e-2) {
                printf("LOSS MISMATCH: %f %f\n", model.mean_loss, *expected_loss);
                allok = 0;
            } else {
                printf("LOSS OK: %f %f\n", model.mean_loss, *expected_loss);
            }
        }
    }

    printf("overall okay: %d\n", allok);

    // free everything
    free(x);
    free(y);
    free(expected_logits);
    free(expected_loss);
    free(expected_grads_memory);
    gpt2_free(&model);
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));

    return 0;
}