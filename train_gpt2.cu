#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net trained in raw CUDA
*/

#include <stdio.h>
#include <stdlib.h>
#ifdef _WIN32
#define _USE_MATH_DEFINES
#endif
#include <math.h>
#include <string.h>
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "platform_utils.h"

// ----------------------------------------------------------------------------
// CUDA utils

// convenience macro for calculating grid/block dimensions for kernels
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// CUDA error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// cuBLAS error checking
void cublasCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("[cuBLAS ERROR]: %d %s %d\n", status, file, line);
        exit(EXIT_FAILURE);
    }
}
#define cublasCheck(status) { cublasCheck((status), __FILE__, __LINE__); }

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
hipblasHandle_t cublas_handle;
hipblasLtHandle_t cublaslt_handle;

// ----------------------------------------------------------------------------
// all the kernels

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__global__ void encoder_forward_kernel2(float* out,
                               int* inp, float* wte, float* wpe,
                               int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;

        int ix = inp[b * T + t];

        float* out_btc = out + b * T * C + t * C + c;
        float* wte_ix = wte + ix * C + c;
        float* wpe_tc = wpe + t * C + c;
        *out_btc = *wte_ix + *wpe_tc;
    }
}


__global__ void layernorm_forward_kernel3(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }

    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * (__ldcs(x+c) - m);
        __stcs(o+c, n * weight[c] + bias[c]);
    }
}

__global__ void add_bias(float* out, float* bias, int B, int T, int OC) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < B*T*OC; i += stride) {
        int col = i % OC;
        out[i] += bias[col];
    }
}


__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]

    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (n * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * d];
        v[idx] = inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void unpermute_kernel(float* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

__global__ void scale_kernel(float* inp, float scale, int B, int NH, int T) {
    // scales the pre-softmax attention scores by scale
    // and sets the autoregressive locations to -INFINITY
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * T * T) {
        int rest = idx % (NH * T * T);
        rest = rest % (T * T);
        int t2 = rest / T;
        int t = rest % T;
        if (t > t2) {
            inp[idx] = -INFINITY;
        } else {
            inp[idx] *= scale;
        }
    }
}

__global__ void softmax_forward_kernel4(float* out, float* inp, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel3, but can handle any block size (multiple of 32)
    // each row of C elements is handled by block_size threads
    // furthermore, each block_size threads get executed in warps of 32 threads

    // special reduction operations warpReduceMax/warpReduceSum are used for intra-warp reductions
    // shared memory is used for inter-warp reduction
    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // the number of warps per block. recall that blockDim.x is block_size
    int warpsPerBlock = blockDim.x / 32;

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // one row of inp, i.e. inp[idx, :] of shape (C,)
    float* x = inp + idx * C;

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x) {
        maxval = fmaxf(maxval, x[i]);
    }
    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);

    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = expf(x[i] - offset);
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // thread coarsening for sum
    x = out + idx * C;
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x) {
        sumval += x[i];
    }
    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);

    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = x[i] / sum;
    }
}

__global__ void residual_forward_kernel(float* out, float* inp1, float* inp2, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = inp1[idx] + inp2[idx];
    }
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)
__global__ void gelu_kernel(float* out, const float* inp, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float xi = inp[i];
        float cube = 0.044715f * xi * xi * xi;
        out[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
    }
}

__global__ void crossentropy_forward_kernel1(float* losses,
                            float* probs, int* targets,
                            int B, int T, int V) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < B * T) {
        int b = i / T;
        int t = i % T;
        float* probs_bt = probs + b * T * V + t * V;
        int ix = targets[b * T + t];
        losses[b * T + t] = -logf(probs_bt[ix]);
    }
}

// ----------------------------------------------------------------------------
// kernel launchers

void encoder_forward(float* out,
                     int* inp, float* wte, float* wpe,
                     int B, int T, int C) {
    const int N = B * T * C;
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    encoder_forward_kernel2<<<grid_size, block_size>>>(out, inp, wte, wpe, B, T, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward(float* out, float* mean, float* rstd,
                       float* inp, float* weight, float* bias,
                       int B, int T, int C) {
    const int block_size = 1024;
    const int N = B * T;
    const int grid_size = CEIL_DIV(N * 32, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// uses cuBLAS
void matmul_forward_cublas(float* out,
                    float* inp, float* weight, float* bias,
                    int B, int T, int C, int OC) {
    const int sqrt_block_size = 32;

    hipblasHandle_t handle; // cuBLAS context
    hipblasStatus_t stat = hipblasCreate(&handle); // initialize CUBLAS context
    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasCheck(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, B*T, C, &alpha, weight, C, inp, C, &beta, out, OC));

    // and now we still have to add the bias... (ew)
    if (bias != NULL) {
        int block_size = sqrt_block_size * sqrt_block_size;
        int grid_size = CEIL_DIV(OC * B * T, block_size);
        add_bias<<<grid_size, block_size>>>(out, bias, B, T, OC);
        cudaCheck(hipGetLastError());
    }
    hipblasDestroy(handle);
}

// uses cuBLASLt to fuse the bias and gelu. does not work with OC = 50257 (last layer)
// https://docs.nvidia.com/cuda/cublas/#cublasltmatmul
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtSgemm/sample_cublasLt_LtSgemm.cu
void matmul_forward_cublaslt(float* out,
                     float* inp, float* weight, float* bias,
                     int B, int T, int C, int OC) {
    int has_bias = (bias != NULL);

    // check bias alignment
    if(((uintptr_t)bias % 16) != 0) {
        printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
        exit(EXIT_FAILURE);
    }

    int returnedResults = 0;
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayout;
    hipblasLtMatrixLayout_t inputLayout;
    hipblasLtMatrixLayout_t outputLayout;
    hipblasLtMatrixLayout_t biasLayout;
    hipblasLtMatmulHeuristicResult_t heuristic;

    // create the operation descriptor
    hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogueBias = HIPBLASLT_EPILOGUE_BIAS;
    cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, cublas_compute_type, HIP_R_32F));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias, sizeof(epilogueBias)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    // define matrix layouts
    cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, HIP_R_32F, C, OC, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, HIP_R_32F, C, B*T, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, HIP_R_32F, OC, B*T, OC));
    cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, HIP_R_32F, OC, 1, OC));

    // create a preference handle with specified max workspace
    cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
    cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    // find a suitable algorithm
    cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
        weightLayout, inputLayout, outputLayout, outputLayout,
        preference, 1, &heuristic, &returnedResults));
    if (returnedResults == 0) {
        printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d, gelu: %d\n", B, T, C, OC, has_bias);
        exit(EXIT_FAILURE);
    }

    // call the matmul
    const float alpha = 1.0f, beta = 0.0f;
    cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
        &alpha, weight, weightLayout, inp, inputLayout, &beta,
        out, outputLayout, out, outputLayout, &heuristic.algo,
        cublaslt_workspace, cublaslt_workspace_size, 0));

    // cleanups
    cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
    cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
    cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}

void attention_forward(float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                       float* inp,
                       int B, int T, int C, int NH) {
    const int block_size = 512;
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    stat = hipblasSgemmStridedBatched(handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            T, T, HS,
                            &alpha,
                            k, HS, T * HS,
                            q, HS, T * HS,
                            &beta,
                            preatt, T, T * T,
                            B * NH);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm failed\n");
        exit(1);
    }

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    total_threads = B * NH * T * T;
    num_blocks = CEIL_DIV(total_threads, block_size);
    scale_kernel<<<num_blocks, block_size>>>(preatt, scale, B, NH, T);

    // softmax. preatt is (B, NH, T, T) but we view it as (B * NH * T, T) and use the softmax kernel
    int softmax_block_size = 256;
    int grid_size = B * NH * T;
    size_t shared_mem_size = 2 * softmax_block_size / 32 * sizeof(float);
    softmax_forward_kernel4<<<grid_size, softmax_block_size, shared_mem_size>>>(att, preatt, B * NH * T, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    stat = hipblasSgemmStridedBatched(handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, T, T,
                            &alpha,
                            v, HS, T * HS,
                            att, T, T * T,
                            &beta,
                            vaccum, HS, T * HS,
                            B * NH);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm failed\n");
        exit(1);
    }

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);

    // cleanups
    hipblasDestroy(handle);
}

void residual_forward(float* out, float* inp1, float* inp2, int N) {
    const int block_size = 256;
    const int grid_size = CEIL_DIV(N, block_size);
    residual_forward_kernel<<<grid_size, block_size>>>(out, inp1, inp2, N);
    cudaCheck(hipGetLastError());
}


void gelu_forward(float* out, const float* inp, int N) {
    const int block_size = 128;
    const int grid_size = CEIL_DIV(N, block_size);
    gelu_kernel<<<grid_size, block_size>>>(out, inp, N);
    cudaCheck(hipGetLastError());
}

void softmax_forward(float* out, float* inp, int N, int C) {
    const int block_size = 256;
    int grid_size = N;
    size_t shared_mem_size = 2 * block_size / 32 * sizeof(float);
    softmax_forward_kernel4<<<grid_size, block_size, shared_mem_size>>>(out, inp, N, C);
}

void crossentropy_forward(float* losses,
                            float* probs, int* targets,
                            int B, int T, int V) {
    const int block_size = 128;
    const int N = B * T;
    const int grid_size = CEIL_DIV(N, block_size);
    crossentropy_forward_kernel1<<<grid_size, block_size>>>(losses, probs, targets, B, T, V);
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------
// GPT-2 model definition

// the parameters of the model
#define NUM_PARAMETER_TENSORS 16
typedef struct {
    float* wte; // (V, C)
    float* wpe; // (maxT, C)
    float* ln1w; // (L, C)
    float* ln1b; // (L, C)
    float* qkvw; // (L, 3*C, C)
    float* qkvb; // (L, 3*C)
    float* attprojw; // (L, C, C)
    float* attprojb; // (L, C)
    float* ln2w; // (L, C)
    float* ln2b; // (L, C)
    float* fcw; // (L, 4*C, C)
    float* fcb; // (L, 4*C)
    float* fcprojw; // (L, C, 4*C)
    float* fcprojb; // (L, C)
    float* lnfw; // (C)
    float* lnfb; // (C)
} ParameterTensors;


// allocate memory for the parameters and point the individual tensors to the right places
float* malloc_and_point_parameters(ParameterTensors* params, size_t* param_sizes, int on_device) {
    // on_device: 0 = CPU, 1 = GPU
    // calculate the number of parameters
    size_t num_parameters = 0;
    for (size_t i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_sizes[i];
    }
    // malloc all parameters all at once on the device
    float* params_memory;
    if (on_device) {
        cudaCheck(hipMalloc((void**)&params_memory, num_parameters * sizeof(float)));
    } else {
        params_memory = (float*)malloc(num_parameters * sizeof(float));
    }
    // assign all the tensors their place in the array
    float** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    float* params_memory_iterator = params_memory;
    for (size_t i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = params_memory_iterator;
        params_memory_iterator += param_sizes[i];
    }
    return params_memory;
}


#define NUM_ACTIVATION_TENSORS 25
typedef struct {
    float* encoded; // (B, T, C)
    float* ln1; // (L, B, T, C)
    float* ln1_mean; // (L, B, T)
    float* ln1_rstd; // (L, B, T)
    float* qkv; // (L, B, T, 3*C)
    float* atty; // (L, B, T, C)
    float* preatt; // (L, B, NH, T, T)
    float* att; // (L, B, NH, T, T)
    float* attproj; // (L, B, T, C)
    float* residual2; // (L, B, T, C)
    float* ln2; // (L, B, T, C)
    float* ln2_mean; // (L, B, T)
    float* ln2_rstd; // (L, B, T)
    float* fch; // (L, B, T, 4*C)
    float* fch_gelu; // (L, B, T, 4*C)
    float* fcproj; // (L, B, T, C)
    float* residual3; // (L, B, T, C)
    float* lnf; // (B, T, C)
    float* lnf_mean; // (B, T)
    float* lnf_rstd; // (B, T)
    float* logits; // (B, T, V)
    float* probs; // (B, T, V)
    float* losses; // (B, T)
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    float* qkvr; // (L, B, T, 3*C)
    float* v_accum; // (L, B, T, C)
} ActivationTensors;

float* malloc_and_point_activations(ActivationTensors* acts, size_t* act_sizes) {
    size_t num_activations = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        num_activations += act_sizes[i];
    }
    float* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, num_activations * sizeof(float)));
    float** ptrs[] = {
        &acts->encoded, &acts->ln1, &acts->ln1_mean, &acts->ln1_rstd, &acts->qkv, &acts->atty,
        &acts->preatt, &acts->att, &acts->attproj, &acts->residual2, &acts->ln2, &acts->ln2_mean,
        &acts->ln2_rstd, &acts->fch, &acts->fch_gelu, &acts->fcproj, &acts->residual3, &acts->lnf,
        &acts->lnf_mean, &acts->lnf_rstd, &acts->logits, &acts->probs, &acts->losses,
        &acts->qkvr, &acts->v_accum
    };
    float* acts_memory_iterator = acts_memory;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        *(ptrs[i]) = acts_memory_iterator;
        acts_memory_iterator += act_sizes[i];
    }
    return acts_memory;
}

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_sizes[NUM_PARAMETER_TENSORS];
    float* params_memory;
    int num_parameters;
    // gradients of the weights
    ParameterTensors grads;
    float* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    // the activations of the model, and their sizes
    ActivationTensors acts;
    size_t act_sizes[NUM_ACTIVATION_TENSORS];
    float* acts_memory;
    int num_activations;
    // gradients of the activations
    ActivationTensors grads_acts;
    float* grads_acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after a forward pass with targets, will be populated with the mean loss
    float* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
} GPT2;


void gpt2_build_from_checkpoint(GPT2 *model, char* checkpoint_path) {

    // read in model from a checkpoint file
    FILE *model_file = fopen(checkpoint_path, "rb");
    if (model_file == NULL) { printf("Error opening model file\n"); exit(1); }
    int model_header[256];
    fread(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file"); exit(1); }
    if (model_header[1] != 1) { printf("Bad version in model file"); exit(1); }

    // read in hyperparameters
    int maxT, V, L, NH, C;
    model->config.max_seq_len = maxT = model_header[2];
    model->config.vocab_size = V = model_header[3];
    model->config.num_layers = L = model_header[4];
    model->config.num_heads = NH = model_header[5];
    model->config.channels = C = model_header[6];
    printf("[GPT-2]\n");
    printf("max_seq_len: %d\n", maxT);
    printf("vocab_size: %d\n", V);
    printf("num_layers: %d\n", L);
    printf("num_heads: %d\n", NH);
    printf("channels: %d\n", C);

    // allocate space for all the parameters and read them in
    model->param_sizes[0] = V * C; // wte
    model->param_sizes[1] = maxT * C; // wpe
    model->param_sizes[2] = L * C; // ln1w
    model->param_sizes[3] = L * C; // ln1b
    model->param_sizes[4] = L * (3 * C) * C; // qkvw
    model->param_sizes[5] = L * (3 * C); // qkvb
    model->param_sizes[6] = L * C * C; // attprojw
    model->param_sizes[7] = L * C; // attprojb
    model->param_sizes[8] = L * C; // ln2w
    model->param_sizes[9] = L * C; // ln2b
    model->param_sizes[10] = L * (4 * C) * C; // fcw
    model->param_sizes[11] = L * (4 * C); // fcb
    model->param_sizes[12] = L * C * (4 * C); // fcprojw
    model->param_sizes[13] = L * C; // fcprojb
    model->param_sizes[14] = C; // lnfw
    model->param_sizes[15] = C; // lnfb

    // cound the number of paramaters
    size_t num_parameters = 0;
    for (size_t i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += model->param_sizes[i];
    }
    printf("num_parameters: %zu\n", num_parameters);
    model->num_parameters = num_parameters;

    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_sizes, 1);

    // read in all the parameters from file and copy them to device
    float* params_memory_cpu = (float*)malloc(num_parameters * sizeof(float));
    fread(params_memory_cpu, sizeof(float), num_parameters, model_file);
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, num_parameters * sizeof(float), hipMemcpyHostToDevice));
    free(params_memory_cpu);
    fclose(model_file);

    // other inits
    model->acts_memory = NULL;
    model->grads_memory = NULL;
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->grads_acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f will designate no loss
}

void gpt2_forward(GPT2 *model, int* inputs, int* targets, int B, int T) {
    // targets are optional and could be NULL

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(1);
    }

    // convenience parameters
    int V = model->config.vocab_size;
    int L = model->config.num_layers;
    int NH = model->config.num_heads;
    int C = model->config.channels;

    // allocate space for all the activations if needed (done here, lazily)
    if(model->acts_memory == NULL) {
        // record the current B,T as well
        model->batch_size = B;
        model->seq_len = T;
        // and now allocate the space
        model->act_sizes[0] = B * T * C; // encoded
        model->act_sizes[1] = L * B * T * C; // ln1
        model->act_sizes[2] = L * B * T; // ln1_mean
        model->act_sizes[3] = L * B * T; // ln1_rstd
        model->act_sizes[4] = L * B * T * 3*C; // qkv
        model->act_sizes[5] = L * B * T * C; // atty
        model->act_sizes[6] = L * B * NH * T * T; // preatt
        model->act_sizes[7] = L * B * NH * T * T; // att
        model->act_sizes[8] = L * B * T * C; // attproj
        model->act_sizes[9] = L * B * T * C; // residual2
        model->act_sizes[10] = L * B * T * C; // ln2
        model->act_sizes[11] = L * B * T; // ln2_mean
        model->act_sizes[12] = L * B * T; // ln2_rstd
        model->act_sizes[13] = L * B * T * 4*C; // fch
        model->act_sizes[14] = L * B * T * 4*C; // fch_gelu
        model->act_sizes[15] = L * B * T * C; // fcproj
        model->act_sizes[16] = L * B * T * C; // residual3
        model->act_sizes[17] = B * T * C; // lnf
        model->act_sizes[18] = B * T; // lnf_mean
        model->act_sizes[19] = B * T; // lnf_rstd
        model->act_sizes[20] = B * T * V; // logits
        model->act_sizes[21] = B * T * V; // probs
        model->act_sizes[22] = B * T; // losses
        model->act_sizes[23] = L * B * T * 3*C; // qkvr
        model->act_sizes[24] = L * B * T * C; // v_accum
        size_t num_activations = 0;
        for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
            num_activations += model->act_sizes[i];
        }
        printf("num_activations: %zu\n", num_activations);
        model->num_activations = num_activations;
        model->acts_memory = malloc_and_point_activations(&model->acts, model->act_sizes);
        // also create memory for caching inputs and targets
        cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
        cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(float)));
    } else {
        // validate B,T is no larger than what was previously allocated
        // in principle, we could re-allocate a larger chunk of memory, for now we just error out
        if (B > model->batch_size || T > model->seq_len) {
            printf("Error: batch size or sequence length is inadequately large\n");
            printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, B, T);
            exit(1);
        }
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    if (targets != NULL) {
        cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    }

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    float* residual;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C); // encoding goes into residual[0]

    for (int l = 0; l < L; l++) {

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        float* l_ln1w = params.ln1w + l * C;
        float* l_ln1b = params.ln1b + l * C;
        float* l_qkvw = params.qkvw + l * 3*C * C;
        float* l_qkvb = params.qkvb + l * 3*C;
        float* l_attprojw = params.attprojw + l * C * C;
        float* l_attprojb = params.attprojb + l * C;
        float* l_ln2w = params.ln2w + l * C;
        float* l_ln2b = params.ln2b + l * C;
        float* l_fcw = params.fcw + l * 4*C * C;
        float* l_fcb = params.fcb + l * 4*C;
        float* l_fcprojw = params.fcprojw + l * C * 4*C;
        float* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        float* l_ln1 = acts.ln1 + l * B * T * C;
        float* l_ln1_mean = acts.ln1_mean + l * B * T;
        float* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        float* l_qkv = acts.qkv + l * B * T * 3*C;
        float* l_qkvr = acts.qkvr + l * B * T * 3*C;
        float* l_atty = acts.atty + l * B * T * C;
        float* l_preatt = acts.preatt + l * B * NH * T * T;
        float* l_att = acts.att + l * B * NH * T * T;
        float* l_v_accum = acts.v_accum + l * B * T * C;
        float* l_attproj = acts.attproj + l * B * T * C;
        float* l_residual2 = acts.residual2 + l * B * T * C;
        float* l_ln2 = acts.ln2 + l * B * T * C;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        float* l_fch = acts.fch + l * B * T * 4*C;
        float* l_fch_gelu = acts.fch_gelu + l * B * T * 4*C;
        float* l_fcproj = acts.fcproj + l * B * T * C;
        float* l_residual3 = acts.residual3 + l * B * T * C;

        // now do the forward pass
        layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C);
        matmul_forward_cublaslt(l_qkv, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        attention_forward(l_atty, l_v_accum, l_qkvr, l_preatt, l_att, l_qkv, B, T, C, NH);
        matmul_forward_cublaslt(l_attproj, l_atty, l_attprojw, l_attprojb, B, T, C, C);
        residual_forward(l_residual2, residual, l_attproj, B*T*C);
        layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C);
        matmul_forward_cublaslt(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, 4*C);
        gelu_forward(l_fch_gelu, l_fch, B*T*4*C);
        matmul_forward_cublaslt(l_fcproj, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C);
        residual_forward(l_residual3, l_residual2, l_fcproj, B*T*C);
    }

    residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    layernorm_forward(acts.lnf, acts.lnf_mean, acts.lnf_rstd, residual, params.lnfw, params.lnfb, B, T, C);
    matmul_forward_cublas(acts.logits, acts.lnf, params.wte, NULL, B, T, C, V);
    softmax_forward(acts.probs, acts.logits, B*T, V);

    // also forward the cross-entropy loss function if we have the targets
    if (targets != NULL) {
        crossentropy_forward(acts.losses, acts.probs, model->targets, B, T, V);

        // for convenience also evaluate the mean loss
        // move the (B,T) losses to CPU
        cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(float), hipMemcpyDeviceToHost));
        float mean_loss = 0.0f;
        for (int i=0; i<B*T; i++) { mean_loss += model->cpu_losses[i]; }
        mean_loss /= B*T;
        model->mean_loss = mean_loss;

    } else {
        // if we don't have targets, we don't have a loss
        model->mean_loss = -1.0f;
    }
}

void gpt2_free(GPT2 *model) {
    cudaCheck(hipFree(model->params_memory));
    cudaCheck(hipFree(model->grads_memory));
    cudaCheck(hipFree(model->m_memory));
    cudaCheck(hipFree(model->v_memory));
    cudaCheck(hipFree(model->acts_memory));
    cudaCheck(hipFree(model->grads_acts_memory));
    cudaCheck(hipFree(model->inputs));
    cudaCheck(hipFree(model->targets));
    hipHostFree(model->cpu_losses);
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip the int main below

// ----------------------------------------------------------------------------
// data loader lite
// returns random batches of data from a file of integers

typedef struct {
    // hyperparameters
    int B;
    int T;
    // input handling and its state
    FILE* tokens_file;
    long file_size;
    long current_position;
    // output memory
    int* batch;
    int* inputs;
    int* targets;
    // convenience variables
    int num_batches;
} DataLoader;

void dataloader_init(DataLoader *loader, char* filename, int B, int T) {
    loader->B = B;
    loader->T = T;

    // open the input file for reading
    loader->tokens_file = fopen(filename, "rb");
    if (loader->tokens_file == NULL) {
        printf("Error opening tokens file\n");
        exit(1);
    }

    // determine the file size
    fseek(loader->tokens_file, 0, SEEK_END);
    loader->file_size = ftell(loader->tokens_file);
    fseek(loader->tokens_file, 0, SEEK_SET);
    if (loader->file_size < (B * T + 1) * sizeof(int)) {
        printf("Error: file size is too small for the batch size and sequence length\n");
        exit(1);
    }
    loader->current_position = 0; // start at the beginning

    // allocate space for B*T + 1 integers to store the inputs and targets
    // Using CUDA CPU pinned memory for faster PCI Express transfers to GPU
    // See: https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
    hipHostMalloc((void**)&loader->batch, (B * T + 1) * sizeof(int));
    loader->inputs = loader->batch;
    loader->targets = loader->batch + 1; // targets are shifted by one
    loader->num_batches = loader->file_size / (B * T * sizeof(int));
}

void dataloader_reset(DataLoader *loader) {
    loader->current_position = 0;
}

void dataloader_next_batch(DataLoader *loader) {
    int B = loader->B;
    int T = loader->T;
    // if we are at the end of the file, loop back to the beginning
    if (loader->current_position + (B*T+1) * sizeof(int) > loader->file_size) {
        loader->current_position = 0;
    }
    // read the B*T+1 integers from the file into batch
    fseek(loader->tokens_file, loader->current_position, SEEK_SET);
    fread(loader->batch, sizeof(int), B*T+1, loader->tokens_file);
    // advance the current position by B*T integers
    loader->current_position += B*T * sizeof(int);
}

void dataloader_free(DataLoader *loader) {
    fclose(loader->tokens_file);
    hipHostFree(loader->batch);
}


// ----------------------------------------------------------------------------
// sampler

#define GPT2_EOT 50256

unsigned int random_u32(unsigned long long *state) {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    *state ^= *state >> 12;
    *state ^= *state << 25;
    *state ^= *state >> 27;
    return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
float random_f32(unsigned long long *state) { // random float32 in [0,1)
    return (random_u32(state) >> 8) / 16777216.0f;
}

int sample_mult(float* probabilities, int n, float coin) {
    // sample index from probabilities (they must sum to 1!)
    // coin is a random number in [0, 1), usually from random_f32()
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += probabilities[i];
        if (coin < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

// ----------------------------------------------------------------------------
// main training loop
int main() {

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("[System]\n");
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // build the GPT-2 model from a checkpoint
    GPT2 model;
    gpt2_build_from_checkpoint(&model, "gpt2_124M.bin");

    // build the DataLoaders from tokens files. for now use tiny_shakespeare if available, else tiny_stories
    char* tiny_stories_train = "data/TinyStories_train.bin";
    char* tiny_stories_val = "data/TinyStories_val.bin";
    char* tiny_shakespeare_train = "data/tiny_shakespeare_train.bin";
    char* tiny_shakespeare_val = "data/tiny_shakespeare_val.bin";
    char* train_tokens = file_exists(tiny_shakespeare_train) ? tiny_shakespeare_train : tiny_stories_train;
    char* val_tokens = file_exists(tiny_shakespeare_val) ? tiny_shakespeare_val : tiny_stories_val;
    int B = 4;
    int T = 1024;
    DataLoader train_loader;
    dataloader_init(&train_loader, train_tokens, B, T);
    printf("train dataset num_batches: %d\n", train_loader.num_batches);
    DataLoader val_loader;
    dataloader_init(&val_loader, val_tokens, B, T);
    printf("val dataset num_batches: %d\n", val_loader.num_batches);
    int val_num_batches = 10;
    printf("batch size: %d\n", B);
    printf("sequence length: %d\n", T);
    printf("val_num_batches: %d\n", val_num_batches);

    // some memory for generating samples from the model
    unsigned long long rng_state = 1337;
    const int gen_max_length = 64;
    int gen_tokens[gen_max_length];
    float* cpu_probs = (float*)malloc(model.config.vocab_size * sizeof(float));

    // train
    for (int step = 0; step <= 40; step++) {

        // once in a while estimate the validation loss
        if (step % 10 == 0) {
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                gpt2_forward(&model, val_loader.inputs, val_loader.targets, B, T);
                val_loss += model.mean_loss;
            }
            val_loss /= val_num_batches;
            printf("val loss %f\n", val_loss);
        }

        // once in a while do model inference to print generated text
        if (step > 0 && step % 20 == 0) {
            gen_tokens[0] = GPT2_EOT; // the GPT-2 EOT token kicks off the generation
            for (int t = 1; t < gen_max_length; t++) {
                // note that inference is wasteful here because
                // for each t, we re-compute all activations between 0 and t
                // leaving this alone because you want separate code for inference anyway
                // the inference here is just for sanity checking purposes
                gpt2_forward(&model, gen_tokens, NULL, 1, t);
                float* probs = model.acts.probs + (t-1) * model.config.vocab_size;
                float coin = random_f32(&rng_state);
                // move probs back to CPU and sample
                cudaCheck(hipMemcpy(cpu_probs, probs, model.config.vocab_size * sizeof(float), hipMemcpyDeviceToHost));
                int next_token = sample_mult(cpu_probs, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
            }
            printf("generated: ");
            for (int t = 0; t < gen_max_length; t++) {
                printf("%d ", gen_tokens[t]);
            }
            printf("\n");
        }

        // do a training step
        const double start_time_ms = get_time_ms();
        dataloader_next_batch(&train_loader);
        gpt2_forward(&model, train_loader.inputs, train_loader.targets, B, T);
        // these are still TODO
        // gpt2_zero_grad(&model);
        // gpt2_backward(&model);
        // gpt2_update(&model, 1e-4f, 0.9f, 0.999f, 1e-8f, 0.0f, step+1);
        cudaCheck(hipDeviceSynchronize()); // finish all CUDA work to get correct precise timings
        const double end_time_ms = get_time_ms();
        const double time_elapsed_ms = end_time_ms - start_time_ms;
        printf("step %d: train loss %f (took %f ms)\n", step, model.mean_loss, time_elapsed_ms);
    }

    // free
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    gpt2_free(&model);
    free(cpu_probs);
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));

    return 0;
}
#endif