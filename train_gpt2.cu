#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net trained in raw CUDA
GPT-2 Transformer Neural Net trained in raw CUDA
Non-trivial notes to be aware of:

We are being clever in the backward pass to conserve memory.
In particular, all parameters use a += in the backward pass, so we
can later do gradient accumulation. But all activations have = instead of +=
because these are faster (just read, no write). This is okay for all activations
except for those in the residual stream, where the gradients have to add. We make
sure that those parts work out ok and that we do a += as necessary. E.g.,
the layernorms are connected to the residuals so we += in layernorm backward.

In this file we are using Mixed Precision training, so different activations,
parameters, grads and buffers may be kept at different precisions, to take
advantage of the fast low-precision hardware in the latest GPUs (bf16/fp16),
and fp8 (coming soon^TM).

Compile:
make train_gpt2cu

Example launch using bfloat16 on 1 GPU batch size 8, sample/eval every 200 steps:
Also we're using TinyStories here for example as it is a bigger dataset
./train_gpt2cu -b 8 -v 200 -s 200 -i data/TinyStories

Example launch using bfloat16 on 4 GPUs, same as above:
mpirun -np 4 ./train_gpt2cu -b 8 -v 200 -s 200 -i data/TinyStories

If you'd like to see train_gpt2.cu produce identical results to
`python train_gpt2.py`, you can run it like this:
make train_gpt2cu && ./train_gpt2cu -b 4 -t 64 -l 1e-4 -v 200 -s 200 -a 1 -x 10 -f 0
make train_gpt2cu PRECISION=FP32 && ./train_gpt2cu -b 4 -t 64 -l 1e-4 -v 200 -s 200 -a 1 -x 10 -f 0
This reads & runs in fp32, B=4, T=64, LR=1e-4, val/sample never (200),
-a 1 is "overfit single batch", -x 10 is 10 iterations, and -f 0 disables tf32
*/

#include <unistd.h>
#include <stdio.h>
#include <stdarg.h>
#include <string>
#include <vector>
#include <algorithm>
#include <functional>
#include <unordered_map>
// GPU / CUDA related
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <nvtx3/nvToolsExt.h>
#include <hip/hip_runtime_api.h>
// Multi-GPU related
#ifdef MULTI_GPU
#include <mpi.h>
#include <nccl.h>
#endif
// our own utilities
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
#include "utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "tokenizer.h"
// defines: dataloader_init, dataloader_reset, dataloader_next_batch, dataloader_free
// defines: evalloader_init, evalloader_reset, evalloader_next_batch, evalloader_free
#include "dataloader.h"
// defines: manual_seed, normal_
// numerically identical to PyTorch's torch.manual_seed and torch.normal
#include "rand.h"
// ----------------------------------------------------------------------------
// CUDA precision settings

enum PrecisionMode {
    PRECISION_FP32,
    PRECISION_FP16,
    PRECISION_BF16
};

// Specific configurations based on the enabled precision
#if defined(ENABLE_FP32)
typedef float floatX;
#define CUBLAS_LOWP HIP_R_32F
#define PRECISION_MODE PRECISION_FP32
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclFloat;
#endif

// use fp16 (note: this may require gradient scaler, currently not implemented!)
#elif defined(ENABLE_FP16)
typedef half floatX;
#define CUBLAS_LOWP HIP_R_16F
#define PRECISION_MODE PRECISION_FP16
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclHalf;
#endif

#else // Default to bfloat16
typedef __hip_bfloat16 floatX;
#define CUBLAS_LOWP HIP_R_16BF
#define PRECISION_MODE PRECISION_BF16
#ifdef MULTI_GPU
const ncclDataType_t ncclFloatX = ncclBfloat16;
#endif
#endif

// ----------------------------------------------------------------------------
// CUDA utils

// Profiler utils
class NvtxRange {
 public:
    NvtxRange(const char* s) { nvtxRangePush(s); }
    NvtxRange(const std::string& base_str, int number) {
        std::string range_string = base_str + " " + std::to_string(number);
        nvtxRangePush(range_string.c_str());
    }
    ~NvtxRange() { nvtxRangePop(); }
};
#define NVTX_RANGE_FN() NvtxRange nvtx_range(__FUNCTION__)

// try to make sure that 2 blocks fit on A100/H100 to maximise latency tolerance
// this needs to be defines rather than queried to be used for __launch_bounds__
#if __CUDA_ARCH__ == 800 || __CUDA_ARCH__ >= 900
#define MAX_1024_THREADS_BLOCKS 2
#else
#define MAX_1024_THREADS_BLOCKS 1
#endif

// WarpSize is not a compile time constant, this allows the compiler to optimize
#define WARP_SIZE 32U

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
const size_t cublaslt_workspace_size = 32 * 1024 * 1024;
void* cublaslt_workspace = NULL;
hipblasComputeType_t cublas_compute = HIPBLAS_COMPUTE_32F;
hipblasLtHandle_t cublaslt_handle;
hipblasHandle_t cublas_handle;
hipDeviceProp_t deviceProp;

// convenience macro for calculating grid/block dimensions for kernels
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// CUDA error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// cuBLAS error checking
void cublasCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("[cuBLAS ERROR]: %d %s %d\n", status, file, line);
        exit(EXIT_FAILURE);
    }
}
#define cublasCheck(status) { cublasCheck((status), __FILE__, __LINE__); }

#ifdef MULTI_GPU
void nccl_check(ncclResult_t status, const char *file, int line) {
    if (status != ncclSuccess) {
        printf("[NCCL ERROR] at file %s:%d:\n%s\n", file, line, ncclGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}
#define ncclCheck(err) (nccl_check(err, __FILE__, __LINE__))

void mpi_check(int status, const char *file, int line) {
    if (status != MPI_SUCCESS) {
        char mpi_error[4096];
        int mpi_error_len = 0;
        assert(MPI_Error_string(status, &mpi_error[0], &mpi_error_len) == MPI_SUCCESS);
        printf("[MPI ERROR] at file %s:%d:\n%.*s\n", file, line, mpi_error_len, mpi_error);
        exit(EXIT_FAILURE);
    }
}
#define mpiCheck(err) (mpi_check(err, __FILE__, __LINE__))
#endif

// older nvcc does not provide __ldcs and __stcs for bfloat16, despite these actually just being unsigned shorts.
// we need to be careful here to only define our own versions if none already exist, otherwise the compiler will
// complain.
// If not, you easily get "no viable overload" (for sm52) and "function already exists" (sm_80)
#if defined(ENABLE_BF16) && (__CUDACC_VER_MAJOR__ < 12) && !((__CUDA_ARCH__ >= 800) || !defined(__CUDA_ARCH__))
__device__ floatX __ldcs(const floatX* address) {
    unsigned short bf = __ldcs(reinterpret_cast<const unsigned short*>(address));
    return __hip_bfloat16_raw{bf};
}

__device__ void __stcs(floatX* address, floatX value) {
    __stcs(reinterpret_cast<unsigned short*>(address), ((__hip_bfloat16_raw)value).x);
}
#endif

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}
// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_xor_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}
// requires all 32 threads in the warp to be active, but should work for any block size
// uses non-dynamic shared memory so every call increases shared memory requirements by 128 bytes
// the fact it's unique shared memory allows us to avoid an extra __syncthreads() call at the end
// but if called inside a loop, the shared memory will be implicitly reused, so set final_sync to 1
using reduction_func_t = float (*) (float);
template<reduction_func_t warp_reduction>
__device__ float blockReduce(float val, bool final_sync=false, float out_of_bounds=0.0f) {
    // two reductions of up to 1024 threads:
    // 1) inside warp (shuffle), 2) cross-warp (shared memory), 3) inside warp (shuffle)
    __shared__ float shared_val[WARP_SIZE];
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int num_warps = blockDim.x / WARP_SIZE;

    float warp_val = warp_reduction(val);
    if (lane_id == 0) { shared_val[warp_id] = warp_val; }
    __syncthreads();
    warp_val = (lane_id < num_warps) ? shared_val[lane_id] : out_of_bounds;
    float block_val = warp_reduction(warp_val);

    if (final_sync) {
        __syncthreads(); // only needed in loops when effectively reusing shared memory etc.
    }
    return block_val;
}

// ----------------------------------------------------------------------------
// Packed128 data structure, which forces the compiler to use 128-bit loads/stores
// in GPUs that support (the LDG.128 and STS.128 instructions)
// This is a bit similar to the use of float4 in the case of 32-bit floats, but
// supports arbitrary precision.

template<class ElementType>
struct alignas(16) Packed128 {
    Packed128() = default;
    __device__ explicit Packed128(int4 bits) {
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&payload, &bits, sizeof(bits));
    }
    __device__ ElementType& operator[](int index) {
        return payload[index];
    }
    __device__ const ElementType& operator[](int index) const {
        return payload[index];
    }
    __device__ int4 get_bits() const {
        int4 bits;
        static_assert(sizeof(bits) == sizeof(payload), "Size mismatch.");
        memcpy(&bits, &payload, sizeof(bits));
        return bits;
    }
    static constexpr const size_t size = sizeof(int4) / sizeof(ElementType);
    ElementType payload[size];
};

// load a Packed128 from an aligned memory address
template<class ElementType>
__device__ Packed128<ElementType> load128(const ElementType* address) {
    return Packed128<ElementType>{*reinterpret_cast<const int4*>(address)};
}
// load a Packed128 from an aligned memory address with streaming cache hint
template<class ElementType>
__device__ Packed128<ElementType> load128cs(const ElementType* address) {
    return Packed128<ElementType>{__ldcs(reinterpret_cast<const int4*>(address))};
}
// store a Packed128 to an aligned memory address
template<class ElementType>
__device__ void store128(ElementType* target, Packed128<ElementType> value) {
    *reinterpret_cast<int4*>(target) = value.get_bits();
}
// store a Packed128 to an aligned memory address with streaming cache hint
template<class ElementType>
__device__ void store128cs(ElementType* target, Packed128<ElementType> value) {
    __stcs(reinterpret_cast<int4*>(target), value.get_bits());
}
// store a Packed128 to an aligned memory address while caching in L2 but bypassing L1
template<class ElementType>
__device__ void store128cg(ElementType* target, Packed128<ElementType> value) {
    __stcg(reinterpret_cast<int4*>(target), value.get_bits());
}

// short-form typedefs
typedef Packed128<float> f128;
typedef Packed128<floatX> x128;

// ----------------------------------------------------------------------------
// Random Number Generatiom

// Simple xorshift RNG
__device__ __host__ unsigned int random_u32(unsigned long long *state) {
    // xorshift rng: https://en.wikipedia.org/wiki/Xorshift#xorshift.2A
    *state ^= *state >> 12;
    *state ^= *state << 25;
    *state ^= *state >> 27;
    return (*state * 0x2545F4914F6CDD1Dull) >> 32;
}
__device__ __host__ float random_f32(unsigned long long *state) { // random float32 in [0,1)
    return (random_u32(state) >> 8) / 16777216.0f;
}

// SquirrelNoise5 - Squirrel's Raw Noise utilities (version 5)
// This gives us a random number from threadIdx/blockIdx + a single seed for the entire GPU
// todo - possibly overkill and we don't need such high quality random numbers? (tbd)
// http://eiserloh.net/noise/SquirrelNoise5.hpp
__device__ __host__ constexpr unsigned int SquirrelNoise5(int positionX, unsigned int seed)
{
	constexpr unsigned int SQ5_BIT_NOISE1 = 0xd2a80a3f;	// 11010010101010000000101000111111
	constexpr unsigned int SQ5_BIT_NOISE2 = 0xa884f197;	// 10101000100001001111000110010111
	constexpr unsigned int SQ5_BIT_NOISE3 = 0x6C736F4B; // 01101100011100110110111101001011
	constexpr unsigned int SQ5_BIT_NOISE4 = 0xB79F3ABB;	// 10110111100111110011101010111011
	constexpr unsigned int SQ5_BIT_NOISE5 = 0x1b56c4f5;	// 00011011010101101100010011110101
	unsigned int mangledBits = (unsigned int) positionX;
	mangledBits *= SQ5_BIT_NOISE1;
	mangledBits += seed;
	mangledBits ^= (mangledBits >> 9);
	mangledBits += SQ5_BIT_NOISE2;
	mangledBits ^= (mangledBits >> 11);
	mangledBits *= SQ5_BIT_NOISE3;
	mangledBits ^= (mangledBits >> 13);
	mangledBits += SQ5_BIT_NOISE4;
	mangledBits ^= (mangledBits >> 15);
	mangledBits *= SQ5_BIT_NOISE5;
	mangledBits ^= (mangledBits >> 17);
	return mangledBits;
}
__device__ __host__ constexpr unsigned int Get2dNoiseUint(int indexX, int indexY, unsigned int seed)
{
	constexpr int PRIME_NUMBER = 198491317; // Large prime number with non-boring bits
	return SquirrelNoise5(indexX + (PRIME_NUMBER * indexY), seed);
}

// stochastic rounding built on top of Squirel Noise above (with seed updated per step via xorshift)
__device__ __forceinline__ void stochastic_rounding(float in, __hip_bfloat16 *out, unsigned int seed) {
    // todo - is this stochastic rounding *too good*? can we cut any corners?
    unsigned int random = Get2dNoiseUint(threadIdx.x, blockIdx.x, seed);
    unsigned int threshold = random & 0xFFFF;
    unsigned int float_bits = __float_as_uint(in);
    unsigned int rounded_bits = float_bits & 0x0000FFFF;
    float_bits = (rounded_bits > threshold) ? (float_bits | 0xFFFF) : (float_bits  & ~0xFFFF);
    *out = __float2bfloat16_rn(__uint_as_float(float_bits));
}
__device__ __forceinline__ void stochastic_rounding(float in, half *out, unsigned int random) {
    *out = (float)in; // todo - implement this...
}
__device__ __forceinline__ void stochastic_rounding(float in, float *out, unsigned int random) {
    *out = in; // dummy function for when floatX is float (FP32 mode)
}

// ----------------------------------------------------------------------------
// MPI / multi-processing setup

// Parameters specific to training on multiple GPUs.
typedef struct {
    int process_rank;      // Rank of this process among all MPI processes. 0 if no multi-GPU.
    int num_processes;     // Total number of processes. 1 if no multi-GPU.
    int local_device_idx;  // This process GPU index on current machine. 0 if no multi-GPU.

    // Zero Redundancy Optimizer stage - https://fairscale.readthedocs.io/en/stable/deep_dive/oss_sdp_fsdp.html
    // 0-Disabled
    // 1-Optimizer State Sharding (OSS)
    // 2-Optimizer + Gradient State Sharding (SDP)
    // 3-Optimizer + Gradient + Horizontal Model Sharding (FSDP)
    int zero_stage;
    size_t shard_num_parameters;
    size_t shard_offset;
#ifdef MULTI_GPU
    ncclComm_t nccl_comm;  // NCCL communication primitive, used for collective multi-GPU work.
#endif
} MultiGpuConfig;

// one global variable to hold the multi-GPU configuration for this process
MultiGpuConfig multi_gpu_config;

#ifdef MULTI_GPU
// Determine which GPU this process should use.
// Processes on the same machines use different GPU indicies. Processes on other machines don't.
// Copied from NCCL examples: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html#example-2-one-device-per-process-or-thread
int multi_gpu_get_local_device_idx(int process_rank, int num_processes) {
  char hostname[1024];
  hostname[1023] = '\0';
  // All processes on the same machine will share the same hostname.
  gethostname(hostname, 1023);
  for (int i=0; i < 1024; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        break;
    }
  }
  uint64_t hostname_hash = 5381;
  for (int c = 0; hostname[c] != '\0'; c++){ hostname_hash = ((hostname_hash << 5) + hostname_hash) ^ hostname[c]; }

  // Distribute all hostname hashes to all processes.
  uint64_t* all_hostsname_hashes = (uint64_t*)malloc(num_processes * sizeof(uint64_t));
  all_hostsname_hashes[process_rank] = hostname_hash;
  mpiCheck(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, all_hostsname_hashes, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

  // Identify which GPU we need to use.
  int local_device_idx = 0;
  for (int current_process = 0; current_process < num_processes; ++current_process) {
     if (current_process == process_rank) {
      // Found my gpu, local_device_idx now has my target GPU index.
      break;
     }
     if (all_hostsname_hashes[current_process] == all_hostsname_hashes[process_rank]) {
      // This process ID runs on the same machine, but it's not me, skip this GPU
      local_device_idx++;
     }
  }

  free(all_hostsname_hashes);
  return local_device_idx;
}
#endif

MultiGpuConfig multi_gpu_config_init(int *argc, char ***argv) {
#ifdef MULTI_GPU
    // Initialize MPI.
    MultiGpuConfig result;
    mpiCheck(MPI_Init(argc, argv));
    mpiCheck(MPI_Comm_rank(MPI_COMM_WORLD, &result.process_rank));
    mpiCheck(MPI_Comm_size(MPI_COMM_WORLD, &result.num_processes));
    result.local_device_idx = multi_gpu_get_local_device_idx(result.process_rank, result.num_processes);
    cudaCheck(hipSetDevice(result.local_device_idx));
    ncclUniqueId nccl_id;
    if (result.process_rank == 0) {
        ncclCheck(ncclGetUniqueId(&nccl_id));
    }
    mpiCheck(MPI_Bcast((void *)&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD));
    ncclCheck(ncclCommInitRank(&result.nccl_comm, result.num_processes, nccl_id, result.process_rank));
    return result;
#else
    printf("Multi-GPU support is disabled. Using a single GPU.\n");
    cudaCheck(hipSetDevice(0));
    MultiGpuConfig result;
    result.process_rank = 0;
    result.num_processes = 1;
    result.local_device_idx = 0;
    return result;
#endif
}

void multi_gpu_config_free(const MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    ncclCheck(ncclCommDestroy(multi_gpu_config->nccl_comm));
    mpiCheck(MPI_Finalize());
#endif
}

// convenience function that only prints if the rank of process is zero
void printf0(const char *format, ...) {
    if (multi_gpu_config.process_rank == 0) {
        va_list args;
        va_start(args, format);
        vprintf(format, args);
        va_end(args);
    }
}

void set_zero_configs(MultiGpuConfig* multi_gpu_config, int zero_stage, size_t total_parameters) {

    multi_gpu_config->zero_stage = 0;
    multi_gpu_config->shard_num_parameters = total_parameters;
    multi_gpu_config->shard_offset = 0;

    // Check the Zero Stage and define sharding parameters
    if (zero_stage == 0) {
        printf0("| Zero Optimization is disabled                                              |\n");
    }
    else if (zero_stage == 1) {
        if (total_parameters % multi_gpu_config->num_processes != 0) {
            printf0("| Zero Optimization is disabled, Can't equally partition parameters          |\n");
            multi_gpu_config->zero_stage = 0;
        }
        else {
            printf0("| Zero Stage1 is enabled                                                     |\n");
            multi_gpu_config->zero_stage = 1;
            multi_gpu_config->shard_num_parameters = total_parameters / multi_gpu_config->num_processes;
            multi_gpu_config->shard_offset = multi_gpu_config->process_rank * multi_gpu_config->shard_num_parameters;
        }
    }
    else{
        printf0("| Disabling Zero Optimization, Zero Stage2 and Stage3 are not yet supported  |\n");
        multi_gpu_config->zero_stage = 0;
    }
}

// ----------------------------------------------------------------------------
// cuDNN path
#ifdef ENABLE_CUDNN
// functions defined in cudnn_att.cu
void create_cudnn();
void destroy_cudnn();
void attention_forward_cudnn(floatX* out,  // output: (B, T, NH, HS)
                             float* stats, // output for backward pass: (B, NH, T)
                             floatX* inp,  // input: (B, T, 3, NH, HS) QKV
                             int B, int T, int NH, int C);

void attention_backward_cudnn(floatX* dqkvr,                                       // output
                              floatX* dout, floatX* qkvr, floatX* o, float* stats, // inputs
                              int B, int T, int NH, int C);
#else
void create_cudnn() {}
void destroy_cudnn() {}
#endif // ENABLE_CUDNN

// ----------------------------------------------------------------------------
// all the kernels

__global__ void encoder_forward_kernel3(floatX* out,
                               const int* inp, const floatX* wte, const floatX* wpe,
                               int B, int T, int C) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    int N = B * T * C;
    if (idx >= N) { return; }

    int bt = idx / C;
    int b = bt / T;
    int t = bt % T;
    int c = idx % C;

    int ix = inp[b * T + t];

    floatX* out_btc = out + b * T * C + t * C + c;
    const floatX* wte_ix = wte + ix * C + c;
    const floatX* wpe_tc = wpe + t * C + c;

    x128 packed_out;
    x128 wte128 = load128cs(wte_ix);
    x128 wpe128 = load128cs(wpe_tc);
    for (int k = 0; k < x128::size; k++) {
        packed_out[k] = (floatX)((float)wte128[k] + (float)wpe128[k]);
    }
    store128(out_btc, packed_out);
}

template <int BLOCK_SIZE=256>
__global__ void wte_backward_kernel(floatX* dwte,
                                    const int4* bucket_info, const int* workload_indices, const floatX* dout, const int* inp,
                                    unsigned int seed, int B, int T, int C) {
    // In order to be deterministic, we preprocess the inputs on the cpu into "buckets"
    // Each bucket corresponds to (WARP_SIZE * x128::size) channels for a single vocabulary token
    // Each thread handles x128::size channels, e.g. 256 per warp for BF16
    // Each block handles (BLOCK_SIZE / WARP_SIZE) elements in a single bucket in parallel
    // If a bucket has less than 8 elements, some warps will return immediately
    // If a bucket has more than 8 elements, we will loop over all of them
    // The buckets are sorted on the CPU so the largest buckets start 1st
    int bucket = blockIdx.x;
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int c_per_warp = WARP_SIZE * x128::size;

    int bucket_start_idx = bucket_info[bucket].x;
    int bucket_size = bucket_info[bucket].y;
    int bucket_ix = bucket_info[bucket].z;
    int c = bucket_info[bucket].w * c_per_warp + (lane_id * x128::size);

    // Each thread handles "x128::size" channels, so at fp8, each warp would handle 512 channels
    // If C is not a multiple of this (e.g. 768), some buckets/c_groups cannot use the entire warp
    if (c >= C) { return; }
    // Exit early if this is a small bucket and this warp doesn't have any items to process
    if (warp_id >= bucket_size) { return; }

    float accum[x128::size] = {0.0f};
    __shared__ float accum_shared[x128::size * BLOCK_SIZE];

    for(int item = warp_id; item < bucket_size; item += BLOCK_SIZE/WARP_SIZE) {
        int bt = workload_indices[bucket_start_idx + item];
        int b = bt / T;
        int t = bt % T;

        const floatX* dout_btc = dout + b * T * C + t * C + c;
        x128 packed_inp1 = load128cs(dout_btc);
        for (int k = 0; k < packed_inp1.size; k++) {
            accum[k] += (float)packed_inp1[k];
        }
    }

    if (warp_id != 0) {
        // we accumulate into warp 0, so only the other warps need to write to shared memory
        for (int k = 0; k < x128::size; k++) {
            accum_shared[threadIdx.x + k * BLOCK_SIZE] = accum[k];
        }
        return; // only warp 0 is needed after writing to shared memory
    }

    // Read dwte for warp 0 even if other warps are not finished yet to maximise latency tolerance
    floatX* dwte_ix = dwte + bucket_ix * C + c;
    x128 packed_in_out = load128(dwte_ix);

    // note: threads which have returned are considered synchronised by CUDA so no risk of deadlock
    __syncthreads();

    // Accumulate into warp 0's registers by reading the values of the other warps in shared memory
    for (int i = threadIdx.x+WARP_SIZE; i < min(BLOCK_SIZE, bucket_size*WARP_SIZE); i += WARP_SIZE) {
        for (int k = 0; k < x128::size; k++) {
            accum[k] += accum_shared[i + k * BLOCK_SIZE];
        }
    }

    // Add the result to dwte and write back to global memory (read-modify-write)
    for (unsigned int k = 0; k < x128::size; k++) {
        // We use stochastic rounding to go from FP32 to BF16 but the seed should be deterministic
        stochastic_rounding(accum[k] + (float)packed_in_out[k], &packed_in_out[k], seed + k);
    }
    store128(dwte_ix, packed_in_out);
}

__global__ void wpe_backward_kernel(floatX* dwpe,
                                    const floatX* dout, const int* inp,
                                    int B, int T, int C, unsigned int seed) {
    // Each thread handles x128::size "channel positions", e.g. 256 per warp for BF16
    // For gpt2-124M BF16, C=768 and T=1024, so 3 warps per channel and 3072 warps in total
    // For each "channel position" we sum the gradients for every batch at that C/T element
    // This way each dwte element is only updated once, and the kernel is fully deterministic!
    // The previous kernel was not deterministic, as batches were aggregated with atomicAdd
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;
    if (idx >= T * C) { return; }

    // if C is not a multiple of WARP_SIZE*x128::size, it's OK for some warps to handle multiple t
    int t = idx / C;
    int c = idx % C;
    float accum[x128::size] = {0.0f};

    for (int b = 0; b < B; b++) {
        x128 packed_dout = load128cs(dout + (b * T * C) + (t * C) + c); // will never be read again
        for (int k = 0; k < x128::size; k++) {
            accum[k] += (float)packed_dout[k];
        }
    }

    floatX* dwpe_tc = dwpe + (t * C) + c;
    x128 packed_dwpe = load128(dwpe_tc);
    for (unsigned int k = 0; k < x128::size; k++) {
        // We use stochastic rounding to go from FP32 to BF16 but the seed should be deterministic
        stochastic_rounding(accum[k] + (float)packed_dwpe[k], &packed_dwpe[k], seed + k);
    }
    store128(dwpe_tc, packed_dwpe);
}

__global__ void layernorm_forward_kernel3(floatX* __restrict__ out, floatX* __restrict__ mean, floatX* __restrict__ rstd,
                                    const floatX*  __restrict__ inp, const floatX*  __restrict__ weight,
                                    const floatX* __restrict__ bias, int N, int C) {
    int lane_id = threadIdx.x % WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;
    int num_warps = blockDim.x / WARP_SIZE;

    int idx = blockIdx.x * num_warps + warp_id;
    if(idx >= N) { return; } // guard

    // the row of input that this group of threads is responsible for
    const floatX* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = lane_id; i < C; i += WARP_SIZE) {
        sum += (float)x[i];
    }
    sum = warpReduceSum(sum);
    float m = sum / C;
    if(lane_id == 0 && mean != nullptr) {
        __stcs(mean + idx, (floatX)m);
    }

    // rstd
    sum = 0.0f;
    for (int i = lane_id; i < C; i += WARP_SIZE) {
        float diff = (float)x[i] - m;
        sum += diff * diff;
    }
    sum = warpReduceSum(sum);
    float s = rsqrtf(sum / C + 1e-5f);
    if(lane_id == 0 && rstd != nullptr) {
        __stcs(rstd + idx, (floatX)s);
    }

    // final normalization and scaling by weight/bias
    floatX* o = out + idx * C;
    for (int c = lane_id; c < C; c += WARP_SIZE) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * ((float)__ldcs(x+c) - m);
        __stcs(o+c, (floatX)(n * (float)weight[c] + (float)bias[c]));
    }
}

__global__ void fused_residual_forward_kernel5(floatX* residual, floatX* normed, floatX* mean, floatX* rstd,
                                               const floatX* inp1, const floatX* inp2,
                                               const floatX* weight, const floatX* bias,
                                               int N, int C) {
    assert(blockDim.x == WARP_SIZE);

    // load weights and biases into shared memory
    // do this before we allow any threads to exit!
    extern __shared__ char* params[];
    // load128/store128 sometimes generated multiple instructions when the types here were floatX*, so
    // let's keep everything as x128
    x128* s_weight = reinterpret_cast<x128*>(params);
    x128* s_bias = reinterpret_cast<x128*>(params) + (C / x128::size);
    x128* s_res = reinterpret_cast<x128*>(params) + ((2 + threadIdx.y) * C / x128::size);

    int sidx = (threadIdx.x + WARP_SIZE * threadIdx.y) * x128::size;
    for(int i = sidx; i < C; i += blockDim.y * WARP_SIZE * x128::size) {
        s_weight[i/x128::size] = load128(weight + i);
        s_bias[i/x128::size] = load128(bias + i);
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    if(idx > N) return;

    // adjust pointers to current token
    residual += C * idx;
    normed += C * idx;
    inp1 += C * idx;
    inp2 += C * idx;

    const float eps = 1e-5f;
    float sum = 0.0f;
    for(int c = threadIdx.x * x128::size; c < C; c += WARP_SIZE * x128::size) {
        const x128 in1 = load128cs(inp1 + c);
        const x128 in2 = load128cs(inp2 + c);
        x128 out;
        for(int k = 0; k < x128::size; ++k) {
            out[k] = (float)in1[k] + (float)in2[k];
            sum += (float)out[k];
        }
        store128cs(residual + c, out);
        s_res[c / x128::size] = out;
    }

    sum = warpReduceSum(sum);
    float m = sum / C;
    float v = 0.f;

    for(int c = threadIdx.x * x128::size; c < C; c += WARP_SIZE * x128::size) {
        const x128 res = s_res[c / x128::size];
        for(int k = 0; k < x128::size; ++k) {
            v += ((float)res[k] - m) * ((float)res[k] - m);
        }
    }

    v = warpReduceSum(v) / C;
    float s = rsqrtf(v + eps);

    for(int c = threadIdx.x * x128::size; c < C; c += WARP_SIZE * x128::size) {
        const x128 res = s_res[c / x128::size];
        const x128 w = s_weight[c / x128::size];
        const x128 b = s_bias[c / x128::size];
        x128 out;
        for(int k = 0; k < x128::size; ++k) {
            float n = s * ((float)res[k] - m); // normalized output
            float o = n * (float)w[k] + (float)b[k]; // scale and shift it
            out[k] = o;
        }

        store128cs(normed + c, out);
    }
    // cache the mean and rstd for the backward pass later
    if(threadIdx.x == 0) {
        mean[idx] = m;
        rstd[idx] = s;
    }
}


// inputs floatX, outputs FP32 (for current FP32-only activation path for this WIP)
__global__ void permute_kernel(floatX* q, floatX* k, floatX* v,
                               const floatX* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * NH * N * d) { return; }

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    int b = idx / (NH * N * d);
    int rest = idx % (NH * N * d);
    int nh_ = rest / (N * d);
    rest = rest % (N * d);
    int n = rest / d;
    int d_ = rest % d;
    int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
    q[idx] = __ldcs(&inp[inp_idx]);
    k[idx] = __ldcs(&inp[inp_idx + NH * d]);
    v[idx] = __ldcs(&inp[inp_idx + 2 * (NH * d)]);
}

__global__ void permute_kernel_backward(floatX* dinp,
                                        const floatX* dq, const floatX* dk, const floatX* dv,
                                        int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * NH * N * d) { return; }

    int b = idx / (NH * N * d);
    int rest = idx % (NH * N * d);
    int nh_ = rest / (N * d);
    rest = rest % (N * d);
    int n = rest / d;
    int d_ = rest % d;

    int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
    dinp[inp_idx] = dq[idx];
    dinp[inp_idx + NH * d] = dk[idx];
    dinp[inp_idx + 2 * (NH * d)] = dv[idx];
}

__global__ void unpermute_kernel(floatX* inp, floatX *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)

    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx >= B * NH * N * d) { return; }

    int b = idx / (NH * N * d);
    int rest = idx % (NH * N * d);
    int nh_ = rest / (N * d);
    rest = rest % (N * d);
    int n = rest / d;
    int d_ = rest % d;
    int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
    out[other_idx] = __ldcs(&inp[idx]);
}

__global__ void unpermute_kernel_backward(floatX* dinp, const floatX *dout, int B, int N, int NH, int d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * NH * N * d) { return; }

    int b = idx / (NH * N * d);
    int rest = idx % (NH * N * d);
    int nh_ = rest / (N * d);
    rest = rest % (N * d);
    int n = rest / d;
    int d_ = rest % d;
    int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
    dinp[idx] = (floatX)dout[other_idx];
}

__global__ void softmax_forward_kernel5(floatX* out, float inv_temperature, const floatX* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    int lane_id = threadIdx.x % WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;
    int num_warps = blockDim.x / WARP_SIZE;

    // micro-optimization: we iterate backwards so that
    // after the softmax backward operation completes, the cache retains the
    // part of the matrix close to the upper left corner, which benefits the
    // matmul operation that immediately follows.
    // int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank(); // forward order
    int idx = (gridDim.x - blockIdx.x - 1) * num_warps + warp_id; // backward order
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const floatX* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    const float flt_max = 340282346638528859811704183484516925440.0f; // to avoid including float.h
    float maxval = -flt_max;
    float sumval = 0.0f;

    const floatX* x_aligned = reinterpret_cast<const floatX*>(__builtin_assume_aligned(x, 16));
    for (int i = lane_id; i < pos_by_4; i += WARP_SIZE) {
        float regarray[4];
        for (int k = 0; k < 4; ++k) {
            regarray[k] = (float)x_aligned[4*i + k];
        }
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, regarray[k]);
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * (regarray[k] - maxval));
        }
    }

    if(4*pos_by_4 + lane_id <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, (float)x[4*pos_by_4 + lane_id]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * ((float)x[4*pos_by_4 + lane_id] - maxval));
    }

    float global_maxval = warpReduceMax(maxval);
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = warpReduceSum(sumval);
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = lane_id; i <= own_pos; i += WARP_SIZE) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * ((float)__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, (floatX)(ev * norm));
    }
}

__global__ void residual_forward_kernel(floatX* out, const floatX* inp1, const floatX* inp2) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;

    x128 packed_out;
    x128 packed_inp1 = load128cs(inp1 + idx);
    x128 packed_inp2 = load128cs(inp2 + idx);
    for (int k = 0; k < packed_inp1.size; k++) {
        packed_out[k] = (floatX)((float)packed_inp1[k] + (float)packed_inp2[k]);
    }
    store128(out + idx, packed_out);
}

#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)
__global__ void gelu_forward_kernel2(floatX* out, const floatX* inp) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;

    x128 packed_out;
    x128 packed_inp = load128cs(inp + idx); // load and do not keep in cache
    for(int k = 0; k < packed_inp.size; ++k) {
        float xi = (float)packed_inp[k];
        float cube = 0.044715f * xi * xi * xi;
        packed_out[k] = (floatX)(0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube))));
    }
    // store instead of storecs (without cache streaming) in case it is useful for the
    // data to be in the cache for the next operation after this GeLU
    store128(out + idx, packed_out);
}

__global__ void gelu_backward_kernel(floatX* dinp, const floatX* inp, const floatX* dout) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * x128::size;

    x128 packed_dinp;
    x128 packed_inp = load128cs(inp + idx);
    x128 packed_dout = load128cs(dout + idx);
    for (int k = 0; k < packed_inp.size; ++k) {
        float x = (float)packed_inp[k];
        float cube = 0.044715f * x * x * x;
        float tanh_arg = GELU_SCALING_FACTOR * (x + cube);
        float tanh_out = tanhf(tanh_arg);
        float coshf_out = coshf(tanh_arg);
        float sech_out = 1.0f / (coshf_out * coshf_out);
        float local_grad = 0.5f * (1.0f + tanh_out) + x * 0.5f * sech_out * GELU_SCALING_FACTOR * (1.0f + 3.0f * 0.044715f * x * x);
        packed_dinp[k] = (floatX)(local_grad * (float)packed_dout[k]);
    }
    store128(dinp + idx, packed_dinp);
}

template<typename OutFloat, bool UseAuxBuffer>
__global__ void matmul_backward_bias_kernel9(OutFloat* dbias, const floatX* dout, int B, int T, int OC,
                                             std::bool_constant<UseAuxBuffer>) {
    constexpr const int bdx = 4;
    constexpr const int bdy = WARP_SIZE / bdx;
    assert(blockDim.x == bdx);
    assert(blockDim.y == bdy);

    int warp_d = (int)threadIdx.x;
    int warp_c = (int)threadIdx.y;
    int block_d = (int)threadIdx.z;

    const int OC_per_warp = bdy * x128::size;  // 64 at BF16

    int local_oc = warp_c * x128::size;
    int global_oc = blockIdx.x * OC_per_warp + local_oc;

    int local_bt = warp_d + bdx * block_d;
    int bt_per_block = bdx * blockDim.z;

    float accumulators[x128::size];
    for (int k = 0; k < x128::size; k++) {
        accumulators[k] = 0.0f;
    }

    if(global_oc < OC) {
        // sum up over all bt within registers
        for (int idx = blockIdx.y * bt_per_block + local_bt; idx < B * T; idx += gridDim.y * bt_per_block) {
            x128 packed_dout = load128(dout + global_oc + idx*OC);
            for (int k = 0; k < x128::size; k++) {
                accumulators[k] += (float)packed_dout[k];
            }
        }
    }

    __shared__ float sub_results[x128::size][WARP_SIZE][bdy];

    // reduce within-warp results
    for (int k = 0; k < x128::size; k++) {
        float v = accumulators[k];
        v += __shfl_down_sync(0xffffffff, v, 1, 4);
        v += __shfl_down_sync(0xffffffff, v, 2, 4);
        if(warp_d == 0) {
            sub_results[k][block_d][warp_c] = v;
        }
    }
    __syncthreads();

    // block-wide reductions
    for (int k = block_d; k < x128::size; k += blockDim.z) {
        float a = 0.f;
        for (int r = warp_d; r < blockDim.z; r += bdx) {
            float v = sub_results[k][r][warp_c];
            v += __shfl_down_sync(0xffffffff, v, 1, 4);
            v += __shfl_down_sync(0xffffffff, v, 2, 4);
            a += v;
        }
        if(warp_d == 0 && global_oc < OC) {
            if constexpr (!UseAuxBuffer) {
                dbias[global_oc + k] = (OutFloat)(a + (float)dbias[global_oc + k]);
            } else {
                dbias[global_oc + k + blockIdx.y * OC] = a;
            }
        }
    }
}

__global__ void reduce_add_sum_kernel(floatX* dst, const float* src, size_t n, size_t m) {
    const size_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * f128::size;
    assert(n % x128::size == 0);
    if (idx < n) {
        f128 acc;
        for(int k = 0; k < f128::size; ++k) {
            acc[k] = 0.f;
        }

        for(int l = 0; l < m; ++l) {
            f128 s = load128(src + idx + n * l);
            for(int k = 0; k < f128::size; ++k) {
                acc[k] += s[k];
            }
        }
        for(int k = 0; k < f128::size; ++k) {
            dst[idx + k] = (floatX) ((float)dst[idx + k] + acc[k]);
        }
    }
}

__global__ void __launch_bounds__(512, 2) // todo - any warnings on Turing with only 1024 threads?
    layernorm_backward_kernel9(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,
                                const floatX* dout, const floatX* inp, const floatX* weight,
                                const floatX* mean, const floatX* rstd,
                                int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2*C + 2*block_size + 1
    int warpsInBlock = blockDim.x / WARP_SIZE; //number of warps in block
    int warpId = threadIdx.x / WARP_SIZE; // warp index within a block
    int baseIdx = blockIdx.x * warpsInBlock + warpId;
    int warpThreadIdx = threadIdx.x % WARP_SIZE; // Thread index within the warp
    int warpsInGrid = gridDim.x * warpsInBlock;
    int C_per_iteration = WARP_SIZE * x128::size;
    int iterations_C = CEIL_DIV(C, C_per_iteration);

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;
    float* dbias_tmp_shared = shared + 2 * C;
    float* dweight_tmp_shared = shared + 2 * C + blockDim.x;

    // init shared memory to zero
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    unsigned int *tmp_flag = (unsigned int*)(shared + 2*C + 2*blockDim.x);
    __syncthreads();

    for (int idx = baseIdx; idx < B * T; idx += warpsInGrid) {
        int b = idx / T;
        int t = idx % T;

        const floatX* dout_bt = dout + b * T * C + t * C;
        const floatX* inp_bt = inp + b * T * C + t * C;
        floatX* dinp_bt = dinp + b * T * C + t * C;
        const float mean_bt = (float)mean[b * T + t];
        const float rstd_bt = (float)rstd[b * T + t];

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warpThreadIdx * x128::size; i < C; i += WARP_SIZE * x128::size) {
            x128 dout128_i   = load128(dout_bt + i);
            x128 inp128_i    = load128(inp_bt  + i);
            x128 weight128_i = load128(weight  + i);
            for (int k = 0; k < x128::size; k++) {
                float norm_bti = ((float)inp128_i[k] - mean_bt) * rstd_bt;
                float dnorm_i = (float)weight128_i[k] * (float)dout128_i[k];
                dnorm_mean += dnorm_i;
                dnorm_norm_mean += dnorm_i * norm_bti;
            }
        }
        dnorm_mean = warpReduceSum(dnorm_mean) / C;
        dnorm_norm_mean = warpReduceSum(dnorm_norm_mean) / C;

        // now iterate again and accumulate all the gradients
        // unfortunately we cannot use the same index for x128 arrays and shared memory
        // as atomics can only be 32-bit rather than 128-bit (at least pre-SM90/Hopper)
        // so this would result in an 8-way bank conflict, and kill performance
        // so instead, we use a shared memory friendly index, and reorder before the final write
        for (int i = 0; i < iterations_C; i++) {
            int global_index = (warpThreadIdx * x128::size) + (i * C_per_iteration);
            int shared_index = warpThreadIdx + (i * C_per_iteration);
            if (global_index >= C) {
                break;
            }

            x128 dout128   = load128cs(dout_bt + global_index);
            x128 inp128    = load128cs(inp_bt  + global_index);
            x128 dinp128   = load128(dinp_bt   + global_index);
            x128 weight128 = load128(weight    + global_index);

            for (int x = 0; x < x128::size; x++) {
                float dout_i = (float)dout128[x];
                float norm_bti = ((float)inp128[x] - mean_bt) * rstd_bt;
                float dnorm_i = (float)weight128[x] * dout_i;

                // sum up the gradients for bias and weight across the entire block
                // this is basically a reduction (but only inter-warp, not intra-warp)
                // doing it this way allows us to avoid using atomics while using many warps
                if (warpId != 0) {
                    dbias_tmp_shared[threadIdx.x] = dout_i;
                    dweight_tmp_shared[threadIdx.x] = norm_bti * dout_i;
                }
                __syncthreads();
                if (warpId == 0) {
                    float dbias_tmp = dout_i;
                    float dweight_tmp = norm_bti * dout_i;
                    for (int j = 1; j < warpsInBlock; j++) {
                        dbias_tmp += dbias_tmp_shared[threadIdx.x + j * WARP_SIZE];
                        dweight_tmp += dweight_tmp_shared[threadIdx.x + j * WARP_SIZE];
                    }
                    // gradient contribution to bias (using shared memory friendly index)
                    dbias_shared[shared_index + x*WARP_SIZE] += dbias_tmp;
                    // gradient contribution to weight (using shared memory friendly index)
                    dweight_shared[shared_index + x*WARP_SIZE] += dweight_tmp;
                }
                __syncthreads();

                // gradient contribution to input
                float dval = 0.0f;
                dval += dnorm_i; // term 1
                dval -= dnorm_mean; // term 2
                dval -= norm_bti * dnorm_norm_mean; // term 3
                dval *= rstd_bt; // final scale
                dinp128[x] = (floatX)((float)dinp128[x] + dval);
            }
            // cache in L2 as this is read by the next kernel, but bypass L1 to minimise thrashing
            store128cg(dinp_bt + global_index, dinp128);
        }
    }
    __syncthreads();
    // Each block writes its partial sum to global memory
    // The last block to finish becomes responsible for summing up all the partial sums
    // This is done by atomically incrementing a flag (cleared to 0 before launching the kernel)
    unsigned int* scratchFlag = (unsigned int*)(scratch);
    // Increment scratch pointer by a full cacheline so that everything remains cacheline aligned
    scratch += 32;
    float* scratch_dbias = scratch;
    float* scratch_dweight = scratch + C;
    for(int i = threadIdx.x; i < C; i+= blockDim.x) {
        // Write to global memory in the same "shared memory banking friendly" order
        scratch_dbias[i + 2*C*blockIdx.x] = dbias_shared[i];
        scratch_dweight[i + 2*C*blockIdx.x] = dweight_shared[i];
    }

    // todo - everything below could become a separate kernel for better performance with maybe less code
    // not enough parallelism even inside that single SM... do we need another level of reduction?!
    __syncthreads();
    if (threadIdx.x == 0) {
        *tmp_flag = atomicInc(scratchFlag, gridDim.x);
    }
    __syncthreads();
    if (*tmp_flag == gridDim.x-1) {
        // Reduction of the partial sums by the final block
        for(int i = threadIdx.x * f128::size; i < C; i+= blockDim.x * f128::size) {
            f128 dbias_accum(make_int4(0, 0, 0, 0));
            f128 dweight_accum(make_int4(0, 0, 0, 0));

            for (int read_block_idx = 0; read_block_idx < gridDim.x; read_block_idx++) {
                int offset = i + 2*C*read_block_idx;
                f128 dbias128 = load128(scratch_dbias + offset);
                f128 dweight128 = load128(scratch_dweight + offset);
                for(int k = 0; k < f128::size; k++) {
                    dbias_accum[k] += dbias128[k];
                    dweight_accum[k] += dweight128[k];
                }
            }
            store128(dbias_shared + i, dbias_accum);
            store128(dweight_shared + i, dweight_accum);
        }
        __syncthreads();

        // reorder from atomic/shared memory-friendly index to real global memory index
        // and convert from float/FP32 to floatX/BF16 for the final write
        // this is separate also because it cannot use as many warps as the above (f128 vs x128)
        // todo - if we split this code into another kernel, we could maybe do it at the same time?
        for (int i = warpId; i < iterations_C; i += warpsInBlock) {
            int global_index = (warpThreadIdx * x128::size) + (i * C_per_iteration);
            int shared_index = warpThreadIdx + (i * C_per_iteration);
            if (global_index >= C) {
                break;
            }

            x128 dbias128 = load128(dbias + global_index);
            x128 dweight128 = load128(dweight + global_index);
            for (int x = 0; x < x128::size; x++) {
                float s_db = dbias_shared[shared_index + x*WARP_SIZE];
                float s_dw = dweight_shared[shared_index + x*WARP_SIZE];
                dbias128[x] = (floatX)(s_db + (float)dbias128[x]);
                dweight128[x] = (floatX)(s_dw + (float)dweight128[x]);
            }
            store128(dbias + global_index, dbias128);
            store128(dweight + global_index, dweight128);
        }
    }
}

__global__ void softmax_autoregressive_backward_kernel(floatX* dpreatt, const floatX* datt, const floatX* att,
                                                       int B, int T, int C, float scale) {
    constexpr const int BlockSize = 256;
    constexpr int T_per_block = 4;

    // go through blocks in reverse order, so the slowest block starts first
    int t0 = T - 1 - T_per_block*blockIdx.x;
    int idx = blockIdx.y;

    att += idx * T * T;
    datt += idx * T * T;
    dpreatt += idx * T * T;

    for(int to = 0; to < T_per_block; ++to) {
        int t = t0 - to;
        if(t < 0) return;
        const floatX* att_bth = att + t * T;
        const floatX* datt_bth = datt + t * T;
        floatX* dpreatt_bth = dpreatt + t * T;

        float local_sum = 0;
        for (int t2 = threadIdx.x; t2 <= t; t2 += BlockSize) {
            local_sum += (float)att_bth[t2] * (float)datt_bth[t2];
        }

        local_sum = blockReduce<warpReduceSum>(local_sum);

        for (int t3 = threadIdx.x; t3 <= t; t3 += BlockSize) {
            // don't touch the cache. Some parts will still be here from the previous loop, and
            // we want to exploit those.
            float acc = (float)__ldcs(att_bth + t3) * ((float)__ldcs(datt_bth + t3) - local_sum);
            __stcs(dpreatt_bth + t3, (floatX)(scale * acc));
        }
    }
}

// Implements linear interpolation using only two floating-point operations (as opposed to three in a naive implementation).
// Reference: https://developer.nvidia.com/blog/lerp-faster-cuda
__device__ float lerp(float start, float end, float weight) {
    return fma(weight, end, fma(-weight, start, start));
}

template <typename Tp, typename Tg>
__global__ void adamw_kernel3(Tp* params_memory, float* master_params_memory, Tg* grads_memory, float* m_memory, float* v_memory,
                              float beta1, float beta2, float learning_rate_inv_beta1_correction, float inv_beta2_correction, float eps, float learning_rate_weight_decay,
                              float grad_scale, unsigned int seed) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // get the gradient, m, and v for this parameter
    float grad = grad_scale * (float)grads_memory[idx];
    float m = m_memory[idx];
    float v = v_memory[idx];
    // update the first moment (momentum)
    m = lerp(grad, m, beta1);
    m_memory[idx] = m;
    // update the second moment (RMSprop)
    v = lerp(grad * grad, v, beta2);
    v_memory[idx] = v;
    m *= learning_rate_inv_beta1_correction;
    v *= inv_beta2_correction;
    // fetch the old value of this parameter as a float, from either source
    float old_param = (master_params_memory != NULL) ? master_params_memory[idx] : (float)params_memory[idx];
    // update this parameter
    float param = old_param - (m / (sqrtf(v) + eps)) + (learning_rate_weight_decay * old_param);
    // update our low precision version of the parameters using stochastic rounding
    // this will be used in the next forward pass
    // TODO: simply doing `params_memory[i] = (floatX)param;` breaks everything (why?)
    unsigned int random = Get2dNoiseUint(threadIdx.x, blockIdx.x, seed);
    stochastic_rounding(param, &params_memory[idx], random);
    // write the full, float version of the param into our master copy, if we maintain one
    // this will be used in the next update
    if (master_params_memory != NULL) { master_params_memory[idx] = param; }
}

template<class T>
__global__ void global_norm_squared_kernel(float* out, const T* data, size_t count) {
    // we want as few atomics as possible, so each block tries to do
    // the maximum amount of work (so no fixed chunk, but instead iterating
    // until we run out of data), and then we reduce inside the block
    // and finally have just one atomic per block.
    // out will be updated atomically from all thread blocks. It is a float, so the
    // atomic op is unproblematic
    size_t index = threadIdx.x + blockDim.x * blockIdx.x;
    size_t grid_width = blockDim.x * gridDim.x;
    float accumulator = 0.f;
    for(size_t i = index; i < count; i += grid_width) {
        accumulator += (float)data[i] * (float)data[i];
    }
    // warp-level reduce
    float block_sum = blockReduce<warpReduceSum>(accumulator);
    if(threadIdx.x == 0) {
        atomicAdd(out, block_sum);
    }
}

struct SoftmaxParams {
    float Scale;
    float Offset;
};

__device__ SoftmaxParams prepare_softmax_blockwide3(int64_t idx, const floatX* inp, int V, int P) {
    // same but not float4
    // one row of inp, i.e. inp[idx, :] of shape (V,)

    const floatX* x = inp + idx * P;
    float thread_maxval = -INFINITY;
    float thread_sumval = 0.0f;
    int i = (V+x128::size-1)/x128::size + threadIdx.x - blockDim.x;

    // special-case loop to handle the unaligned elements at the end of the array
    // this lets us skip the bounds check in the main loop below, which improves performance
    while ((i+1)*x128::size > V) {
        for(int k = 0; k < x128::size; ++k) {
            if (i*x128::size+k >= V) {
                break; // bounds checking against real V (rather than padded P)
            }
            float v = (float)x[i*x128::size+k];
            float old_maxval = thread_maxval;
            thread_maxval = fmaxf(thread_maxval, v);
            thread_sumval *= expf((old_maxval - thread_maxval));
            thread_sumval += expf(v - thread_maxval);
        }
        i -= blockDim.x;
    }

    // main loop for the bulk of the iterations (no bounds checking required!)
    for (; i >= 0; i -= blockDim.x) {
        x128 packed_x = load128(x + i * x128::size); // load and keep in cache until fused_classifier loop
        for(int k = 0; k < x128::size; ++k) {
            float v = (float)packed_x[k];
            float old_maxval = thread_maxval;
            thread_maxval = fmaxf(thread_maxval, v);
            thread_sumval *= expf((old_maxval - thread_maxval));
            thread_sumval += expf(v - thread_maxval);
        }
    }

    // Block Max Reduction -> Maths -> Block Sum Reduction
    float block_maxval = blockReduce<warpReduceMax>(thread_maxval, false, -INFINITY);
    thread_sumval *= expf(thread_maxval - block_maxval);
    float block_sumval = blockReduce<warpReduceSum>(thread_sumval);

    // return the softmax parameters
    return SoftmaxParams{1.f / block_sumval, block_maxval};
}

// will _update_ logits to logit gradients
// uses template to decide whether to write logits and probs
// split both loops in "multiple-of-x128-size" and "bounds-checked remainder" parts
template <bool WriteLogits = true, bool WriteProbs = false>
__global__ void __launch_bounds__(1024, MAX_1024_THREADS_BLOCKS)
                fused_classifier_kernel5(floatX* logits, floatX* losses, floatX* probs,
                                         const float dloss, const int* targets,
                                         int B, int T, int V, int P) {
    // note: idx is small enough that it easily fits into 32 bit;
    // by making it a long here, we ensure that any offsets calculated with it (e.g., idx * P)
    // are done is 64 bit
    int64_t idx = gridDim.x - (blockIdx.x+1); // reverse order for cache hits on matmul data
    int ix = targets[idx];

    // softmax (reading B * T * V, same logits read again below, hopefully still in cache)
    SoftmaxParams sp = prepare_softmax_blockwide3(idx, logits, V, P);

    // calculate the probability needed for the loss and update (single-threaded)
    if(threadIdx.x == 0) {
        float prob = expf((float)logits[idx * P + ix] - sp.Offset) * sp.Scale;
        losses[idx] = (floatX)(-logf(prob));
    }

    // calculate the gradients directly, saves bandwidth from probs during training
    // but also supports writing probs for inference-only and debugging
    const floatX* logits_vec = logits + idx * P;
    for (int i = threadIdx.x; i < V/x128::size; i += blockDim.x) {
        // this is the 2nd read of logits after the one in prepare_softmax2
        // it will be overwritten by the logits gradients which is when we reduce cache persistence
        x128 packed_logits_vec = load128(logits_vec + i * x128::size); // rely on cs of store128cs
        x128 packed_probs;
        for(int k = 0; k < x128::size; ++k) {
            int element = i*x128::size + k;
            float prob = expf((float)packed_logits_vec[k] - sp.Offset) * sp.Scale;
            packed_probs[k] = (floatX)prob;
            float indicator = (element == ix) ? 1.0f : 0.0f;
            packed_logits_vec[k] = (floatX)((prob - indicator) * dloss);
        }
        if (WriteLogits){
            // reduce cache persistence for the overwritten logits
            // to maximise probability that logits remain in cache between prepare_softmax and here
            store128cs(logits + idx * P + i * x128::size, packed_logits_vec);
        }
        if (WriteProbs) {
            store128(probs + idx * P + i * x128::size, packed_probs);
        }
    }

    // handle remaining elements after the last multiple of x128::size
    // e.g. if V = 8003, and x128::size = 8, we need to handle the last 3 elements
    int unaligned_start = V & ~(x128::size - 1); // round down to multiple of x128::size
    for (int i = threadIdx.x + unaligned_start; i < V; i++) {
        float prob = expf((float)logits_vec[i] - sp.Offset) * sp.Scale;
        float indicator = (i == ix) ? 1.0f : 0.0f;
        float dlogit = (prob - indicator) * dloss;
        if (WriteLogits){
            __stcs(logits + idx * P + i, (floatX)dlogit);
        }
        if (WriteProbs) {
            probs[idx * P + i] = (floatX)prob;
        }
    }
}

// device functions and the kernel to cast data between types
template<typename Td, typename Ts>
__device__ Td cast_value(Ts val);

template<>
__device__ float cast_value<float, float>(float val) {
    return val;
}

template<>
__device__ float cast_value<float, half>(half val) {
    return __half2float(val);
}

template<>
__device__ float cast_value<float, __hip_bfloat16>(__hip_bfloat16 val) {
    return __bfloat162float(val);
}

template<typename Td, typename Ts>
__global__ void copy_and_cast_kernel(Td* dst, const Ts* src, size_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // need to try grid stride looping for more perf later
    if (idx < n) {
        dst[idx] = cast_value<Td, Ts>(src[idx]);
    }
}

// ----------------------------------------------------------------------------
// kernel launchers

void encoder_forward(floatX* out,
                     const int* inp, const floatX* wte, const floatX* wpe,
                     int B, int T, int C) {
    NVTX_RANGE_FN();
    const int block_size = 256;
    const int N = B * T * C;
    const int grid_size = CEIL_DIV(N, (int)(block_size * x128::size));
    encoder_forward_kernel3<<<grid_size, block_size>>>(out, inp, wte, wpe, B, T, C);
    cudaCheck(hipGetLastError());
}

// Fully deterministic (see comments in wte_backward_kernel and wpe_backward_kernel for more details)
void encoder_backward(floatX* dwte, floatX* dwpe, floatX* scratch, // gpu outputs & scratch
                      int* workload_indices, int4* bucket_info,    // cpu scratch buffers
                      const floatX* dout, const int* inp, const int* inputs_cpu, // cpu/gpu inputs
                      int B, int T, int C, unsigned int seed) {
    NVTX_RANGE_FN();

    // Launch wpe kernel first (so it runs on the GPU in parallel with the CPU pre-processing for wte)
    const int block_size = 256;
    const int N = T * C / x128::size;
    const int grid_size = CEIL_DIV(N, block_size);
    wpe_backward_kernel<<<grid_size, block_size, 0>>>(dwpe, dout, inp, B, T, C, seed);

    // check the GPU scratch buffer is large enough to hold the bucket info and workload indices
    // todo - this is trivially true given hardcoded scratch buffer size here, is this useful?
    int num_c_groups = CEIL_DIV(C, x128::size * WARP_SIZE);
    assert(B*T*num_c_groups * (sizeof(int4)+sizeof(int)) <= B*T*3*C * sizeof(floatX));

    // Step 1: Sort inputs into buckets
    int total_items = 0;
    std::unordered_map<uint64_t, std::vector<uint64_t>> buckets;
    for (uint64_t bt = 0; bt < B * T; bt++) {
        for (uint64_t c_group = 0; c_group < num_c_groups; c_group++) {
            // todo - passing c_group/inputs_cpu[bt] in data to avoid a second hash lookup is a bit hacky
            uint64_t data = bt + (c_group<<32ULL) + ((uint64_t)inputs_cpu[bt]<<42ULL);
            buckets[c_group + num_c_groups * inputs_cpu[bt]].push_back(data);
            total_items++;
        }
    }

    // Step 2: Sort buckets by size in descending order
    // this is so the largest buckets are processed first by the GPU
    // otherwise, if they started late, they would still be running with the rest of the GPU idle
    std::vector<std::pair<uint64_t, std::vector<uint64_t>>> sortedBuckets(buckets.begin(), buckets.end());
    std::sort(sortedBuckets.begin(), sortedBuckets.end(), // ugly because we don't have a typedef for the std::pair
              [](const std::pair<uint64_t, std::vector<uint64_t>>& a, const std::pair<uint64_t, std::vector<uint64_t>>& b) {
                  return a.second.size() > b.second.size();
              });

    int num_buckets = buckets.size();
    int bucket_index = 0;
    int workload_index = 0;
    for (const auto& bucket : sortedBuckets) {
        bucket_info[bucket_index].x = workload_index; // bucket start
        bucket_info[bucket_index].y = bucket.second.size(); // bucket size
        bucket_info[bucket_index].z = (bucket.second[0] >> 42ULL) & ((1ULL<<20ULL)-1); // bucket ix
        bucket_info[bucket_index].w = (bucket.second[0] >> 32ULL) & ((1ULL<<10ULL)-1); // bucket c

        for (uint64_t idx : bucket.second) {
            workload_indices[workload_index++] = (int)(idx & ((1ULL<<31ULL)-1ULL));
        }
        bucket_index++;
    }

    // Step 3: Copy data from host to device (async until the last one to avoid synchronising CPU/GPU twice)
    // todo - could use CUDA events (even without streams) to avoid CPU/GPU synchronisation completely
    int4* d_bucket_info = (int4*)scratch;
    int*  d_workload_indices = (int*)(scratch + B*T*num_c_groups * sizeof(int4));
    hipMemcpyAsync(d_bucket_info, bucket_info, num_buckets * sizeof(int4), hipMemcpyHostToDevice);
    hipMemcpy(d_workload_indices, workload_indices, total_items * sizeof(int), hipMemcpyHostToDevice);

    // Launch wte kernel
    // todo - profile block sizes on more content (depends on number of buckets and on GPU?)
    wte_backward_kernel<256><<<num_buckets, 256>>>(dwte, d_bucket_info, d_workload_indices, dout, inp, seed, B, T, C);
    cudaCheck(hipGetLastError());
}

void layernorm_forward(floatX* out, floatX* mean, floatX* rstd,
                       floatX* inp, const floatX* weight, const floatX* bias,
                       int B, int T, int C) {
    NVTX_RANGE_FN();
    const int block_size = 512;
    const int N = B * T;
    const int grid_size = CEIL_DIV(N * WARP_SIZE, block_size);
    layernorm_forward_kernel3<<<grid_size, block_size>>>(out, mean, rstd, inp, weight, bias, N, C);
    cudaCheck(hipGetLastError());
}

// https://docs.nvidia.com/cuda/cublas/#cublasltmatmul
void matmul_forward_cublaslt(floatX* out,
                     floatX* inp, floatX* weight, floatX* bias,
                     int B, int T, int C, int OC) {
    NVTX_RANGE_FN();
    int has_bias = (bias != NULL);

    // check bias alignment
    if(((uintptr_t)bias % 16) != 0) {
        printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
        exit(EXIT_FAILURE);
    }

    // these need to be in FP16 if and only if alpha/beta are HIPBLAS_COMPUTE_16F
    const float alpha = 1.0f, beta = 0.0f;

    int returnedResults = 0;
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayout;
    hipblasLtMatrixLayout_t inputLayout;
    hipblasLtMatrixLayout_t outputLayout;
    hipblasLtMatrixLayout_t biasLayout;
    hipblasLtMatmulHeuristicResult_t heuristic;

    // create the operation descriptor
    hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogueBias = has_bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;

    cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, cublas_compute, HIP_R_32F)); // FP16 if HIPBLAS_COMPUTE_16F
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias, sizeof(epilogueBias)));
    cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    // define matrix layouts
    cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, CUBLAS_LOWP, C, OC, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, CUBLAS_LOWP, C, B*T, C));
    cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, CUBLAS_LOWP, OC, B*T, OC));
    cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, CUBLAS_LOWP, OC, 1, OC));

    // create a preference handle with specified max workspace
    cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
    cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    // find a suitable algorithm
    cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
        weightLayout, inputLayout, outputLayout, outputLayout,
        preference, 1, &heuristic, &returnedResults));
    if (returnedResults == 0) {
        printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d\n", B, T, C, OC, has_bias);
        exit(EXIT_FAILURE);
    }

    // call the matmul
    cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
        &alpha, weight, weightLayout, inp, inputLayout, &beta,
        out, outputLayout, out, outputLayout, &heuristic.algo,
        cublaslt_workspace, cublaslt_workspace_size, 0));

    // cleanups
    cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
    cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
    cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
    cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}

void attention_forward(floatX* out, floatX* qkvr, floatX* att,
                       floatX* inp,
                       int B, int T, int C, int NH) {
    NVTX_RANGE_FN();
    // Note: `inp` is not needed for backward pass, so we re-use it as a scratch buffer.
    // Its contents will be overwritten by this function.
    const int block_size = 256;
    const float alpha = 1.0f, beta = 0.0f;

    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    floatX *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);


    floatX* preatt = inp;
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     T, T, HS, &alpha,
                                     k, CUBLAS_LOWP, HS, T * HS,
                                     q, CUBLAS_LOWP, HS, T * HS,
                                     &beta, preatt, CUBLAS_LOWP, T, T * T,
                                     B * NH, cublas_compute, HIPBLAS_GEMM_DEFAULT));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int grid_size = CEIL_DIV(B * NH * T * 32, block_size);
    softmax_forward_kernel5<<<grid_size, block_size>>>(att, scale, preatt, B * NH, T);

    // new approach: first cuBLAS another batched matmul
    floatX* vaccum = inp;
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     HS, T, T, &alpha,
                                     v, CUBLAS_LOWP, HS, T * HS,
                                     att, CUBLAS_LOWP, T, T * T,
                                     &beta, vaccum, CUBLAS_LOWP, HS, T * HS,
                                     B * NH, cublas_compute, HIPBLAS_GEMM_DEFAULT));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

void residual_forward(floatX* out, const floatX* inp1, const floatX* inp2, int N) {
    NVTX_RANGE_FN();
    const int block_size = 256;
    assert(N % block_size == 0);
    const int grid_size = CEIL_DIV(N, block_size * x128::size);
    residual_forward_kernel<<<grid_size, block_size>>>(out, inp1, inp2);
    cudaCheck(hipGetLastError());
}

void fused_residual_forward5(floatX* residual, floatX* normed, floatX* mean, floatX* rstd,
                             const floatX* inp1, const floatX* inp2,
                             const floatX* weight, const floatX* bias,
                             int N, int C) {
    const int block_size = 256;
    int block_y = block_size / WARP_SIZE;
    const int grid_size = CEIL_DIV(N, block_y);
    size_t smem = (2 + block_y) * C * sizeof(floatX);

    // in order to use more than 48 KiB of smem, need to call hipFuncSetAttribute
    // this may fail, in which case we fall back to the smem free implementation.
    cudaCheck(hipGetLastError());
    auto status = hipFuncSetAttribute(reinterpret_cast<const void*>(fused_residual_forward_kernel5), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
    hipGetLastError();
    if(status == hipSuccess) {
        fused_residual_forward_kernel5<<<grid_size, dim3(WARP_SIZE, block_y), smem>>>(residual, normed, mean, rstd, inp1, inp2,
                                                                               weight, bias, N, C);
    } else {
        residual_forward(residual, inp1, inp2, N*C);
        layernorm_forward(normed, mean, rstd, residual, weight, bias, N, 1, C);
    }
    cudaCheck(hipGetLastError());
}


void gelu_forward(floatX* out, const floatX* inp, int N) {
    NVTX_RANGE_FN();
    const int block_size = 512;
    assert(N % block_size == 0);
    const int grid_size = CEIL_DIV(N, block_size * x128::size);
    gelu_forward_kernel2<<<grid_size, block_size>>>(out, inp);
    cudaCheck(hipGetLastError());
}

void gelu_backward(floatX* dinp, const floatX* inp, const floatX* dout, const int N) {
    NVTX_RANGE_FN();
    const int block_size = 128;
    assert(N % block_size == 0);
    const int grid_size = CEIL_DIV(N, block_size * x128::size);
    gelu_backward_kernel<<<grid_size, block_size>>>(dinp, inp, dout);
    cudaCheck(hipGetLastError());
}

void matmul_backward(floatX* dinp, floatX* dweight, floatX* dbias,
                     floatX* dout, floatX* inp, floatX* weight,
                     float* dbias_buffer,
                     int B, int T, int C, int OC) {
    NVTX_RANGE_FN();
    float one = 1.0f, zero = 0.0f;

    // backward to bias, if given, does a +=
    if (dbias != NULL) {
        // Each warp is responsible for 8 * "x128::size" = 64 OCs at BF16 (OC must be a multiple of 64!)
        // Block size is 1024 | 768 threads (32|24 warps) and we reduce those values into 1 at the end

        const int block_size = deviceProp.maxThreadsPerMultiProcessor == 1536 ? 768 : 1024;

        dim3 block_dim = {4, 8, (unsigned)block_size/WARP_SIZE};
        const int OC_per_warp = block_dim.y * x128::size; // 64 at BF16
        const int grid_size_x = CEIL_DIV(OC, OC_per_warp); // e.g. 12 horizontal blocks for 768 OCs at BF16
        const int grid_size_y = max(1, deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount / (block_size * grid_size_x)); // full GPU!

        // If we have enough OC that we don't need cross-block reductions, we can skip the bias_buffer accumulation
        // and write results directly to the output.
        if(grid_size_y == 1) {
            matmul_backward_bias_kernel9<<<dim3(grid_size_x, grid_size_y), block_dim>>>(dbias, dout, B, T, OC, std::bool_constant<false>{});
            cudaCheck(hipGetLastError());
        } else {
            // kernel 9 overwrites temp buffer, so no need to memset
            matmul_backward_bias_kernel9<<<dim3(grid_size_x, grid_size_y), block_dim>>>(dbias_buffer, dout, B, T, OC, std::bool_constant<true>{});
            cudaCheck(hipGetLastError());
            reduce_add_sum_kernel<<<CEIL_DIV(OC, 256 * f128::size), 256>>>(dbias, dbias_buffer, OC, grid_size_y);
            cudaCheck(hipGetLastError());
        }
    }

    // backward to input, uses = in the backward pass (set the gradient)
    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, C, B*T, OC, &one,
                             weight, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &zero,
                             dinp, CUBLAS_LOWP, C, cublas_compute, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    // backward to weight, uses += in the backward pass (accumulate the gradient) by setting alpha=one
    cublasCheck(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, C, OC, B*T, &one,
                             inp, CUBLAS_LOWP, C, dout, CUBLAS_LOWP, OC, &one,
                             dweight, CUBLAS_LOWP, C, cublas_compute, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    cudaCheck(hipGetLastError());
}

void layernorm_backward(floatX* dinp, floatX* dweight, floatX* dbias, float* scratch,
                        const floatX* dout, const floatX* inp, const floatX* weight, const floatX* mean, const floatX* rstd,
                        int B, int T, int C) {
    NVTX_RANGE_FN();
    const int block_size = 512;
    const int blocks_per_sm = 2; // supported on every architecture and less cache thrashing than 3
    const int grid_size = blocks_per_sm * deviceProp.multiProcessorCount;
    size_t shared_mem_size = (2*C + 2*block_size + 1) * sizeof(float);  // see kernel

    hipMemset(scratch, 0, 1 * sizeof(float)); // only need to reset the flag to 0
    layernorm_backward_kernel9<<<grid_size, block_size, shared_mem_size>>>(dinp, dweight, dbias, scratch, dout, inp, weight, mean, rstd, B, T, C);
    cudaCheck(hipGetLastError());
}


// the sequence of transformations in this compound op is:
// inp (B,T,3C) -> qkvr (B,T,3C) -> preatt (B,NH,T,T) -> att (B,NH,T,T) -> vaccum (B,T,C) -> out (B,T,C)
void attention_backward(floatX* dinp, floatX* dqkvr, floatX* dpreatt, floatX* datt, floatX* scratch,
                        const floatX* dout,
                        const floatX* qkvr, const floatX* att,
                        int B, int T, int C, int NH) {
    NVTX_RANGE_FN();
    const int block_size = 256;
    int HS = C / NH; // head size
    const float alpha = 1.0f, beta = 0.0f;

    // unpack convenience pointers into q, k, v
    const floatX *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    floatX *dq, *dk, *dv;
    dq = dqkvr + 0 * B * T * C;
    dk = dqkvr + 1 * B * T * C;
    dv = dqkvr + 2 * B * T * C;

    // backward through the unpermute operation
    int num_blocks = CEIL_DIV(B * T * C, block_size);
    unpermute_kernel_backward<<<num_blocks, block_size>>>(scratch, dout, B, T, NH, HS);
    // backward into datt
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, T, T, HS, &alpha,
                                           v, CUBLAS_LOWP, HS, T * HS, scratch, CUBLAS_LOWP, HS, T * HS, &beta,
                                           datt, CUBLAS_LOWP, T, T * T, B * NH, cublas_compute, HIPBLAS_GEMM_DEFAULT));
    // backward into dv
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &alpha,
                                           scratch, CUBLAS_LOWP, HS, T * HS, att, CUBLAS_LOWP, T, T * T, &beta,
                                           dv, CUBLAS_LOWP, HS, T * HS, B * NH, cublas_compute, HIPBLAS_GEMM_DEFAULT));
    // backward into preatt
    int hs = C / NH; // head size
    float scale = 1.0f / sqrtf(hs);
    softmax_autoregressive_backward_kernel<<<dim3(T / 4, B * NH), 256, 256>>>(dpreatt, datt, att, B, T, C, scale);
    // backward into q
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HS, T, T, &alpha,
                                           k, CUBLAS_LOWP, HS, T * HS, dpreatt, CUBLAS_LOWP, T, T * T, &beta,
                                           dq, CUBLAS_LOWP, HS, T * HS, B * NH, cublas_compute, HIPBLAS_GEMM_DEFAULT));
    // backward into k
    cublasCheck(hipblasGemmStridedBatchedEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, HS, T, T, &alpha,
                                           q, CUBLAS_LOWP, HS, T * HS, dpreatt, CUBLAS_LOWP, T, T * T, &beta,
                                           dk, CUBLAS_LOWP, HS, T * HS, B * NH, cublas_compute, HIPBLAS_GEMM_DEFAULT));
    // backward into inp
    num_blocks = CEIL_DIV(B * NH * T * HS, block_size);
    permute_kernel_backward<<<num_blocks, block_size>>>(dinp, dq, dk, dv, B, T, NH, HS);
    cudaCheck(hipGetLastError());
}

// replaces logits with logit gradients
template <typename Type>
void fused_classifier(Type* logits, Type* losses,
                      const float dloss, const int* targets,
                      int B, int T, int V, int P) {
    NVTX_RANGE_FN();
    const int block_size = 1024;
    const int N = B * T;
    const int grid_size = N;
    fused_classifier_kernel5<<<grid_size, block_size, 512>>>(logits, losses, (floatX*)NULL, dloss, targets, B, T, V, P);
    cudaCheck(hipGetLastError());
}

template<typename T>
void global_norm_squared(float* out, const T* values, size_t count) {
    const int block_size = 512;
    // launch just enough blocks to fill the grid. deliberately no DIV_CEIL.
    // having one block less than possible is a tiny performance hit, having
    // one block too many is catastrophic, since it only can start once all the other
    // blocks finish. anyway, I think cuda_threads_per_SM should be a multiple of 512
    // on all gpus, so the division really is going to be exact.
    const int grid_size = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount / block_size;
    assert(grid_size > 0);      // gives a better error than letting the call below fail
    // initialize out with zero
    cudaCheck(hipMemset(out, 0, sizeof(float)));
    global_norm_squared_kernel<<<grid_size, block_size>>>(out, values, count);
    cudaCheck(hipGetLastError());
}

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int padded_vocab_size; // padded to e.g. %128==0, 50304
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
constexpr const int NUM_PARAMETER_TENSORS = 16;
typedef struct {
    floatX* wte; // (V, C)
    floatX* wpe; // (maxT, C)
    floatX* ln1w; // (L, C)
    floatX* ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatX* ln2w; // (L, C)
    floatX* ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatX* lnfw; // (C)
    floatX* lnfb; // (C)
} ParameterTensors;
static_assert(sizeof(ParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t C = config.channels;
    size_t maxT = config.max_seq_len;
    size_t L = config.num_layers;
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb

    // populate the parameter sizes in bytes (all the same for now, keeping for future use)
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
}

// allocate memory for the parameters and point the individual tensors to the right places
void* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof) {
    // calculate the total number of parameters and bytes across all tensors
    size_t num_parameters = 0;
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters += param_elements[i];
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    void* params_memory;
    cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

#define NUM_ACTIVATION_TENSORS 21
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    floatX* ln1_mean; // (L, B, T)
    floatX* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    floatX* att; // (L, B, NH, T, T) (smaller with cuDNN)
    floatX* attproj; // (L, B, T, C)
    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    floatX* ln2_mean; // (L, B, T)
    floatX* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* fcproj; // (L, B, T, C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C)
    floatX* lnf_mean; // (B, T)
    floatX* lnf_rstd; // (B, T)
    floatX* losses; // (B, T)
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;
} ActivationTensors;

void fill_in_activation_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config, int recompute) {
    size_t Vp = config.padded_vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    act_sizes[0] = B * T * C; // encoded
    act_sizes[1] = L * B * T * C; // ln1
    act_sizes[2] = L * B * T; // ln1_mean
    act_sizes[3] = L * B * T; // ln1_rstd
    act_sizes[4] = L * B * T * C; // atty
    #ifdef ENABLE_CUDNN
    // FP32 stats tensor for cuDNN to be passed to backward pass
    act_sizes[5] = L * B * NH * T * (sizeof(float) / sizeof(floatX));
    #else
    act_sizes[5] = L * B * NH * T * T; // att
    #endif
    act_sizes[6] = L * B * T * C; // attproj
    act_sizes[7] = L * B * T * C; // residual2
    act_sizes[8] = L * B * T * C; // ln2
    act_sizes[9] = L * B * T; // ln2_mean
    act_sizes[10] = L * B * T; // ln2_rstd
    act_sizes[11] = L * B * T * 4*C; // fch
    // if recompute >= 1 then we will recompute gelu_forward during backward and use this as scratch buffer
    act_sizes[12] = (recompute == 0) ? L * B * T * 4*C : B * T * 4*C;
    act_sizes[13] = L * B * T * C; // fcproj
    act_sizes[14] = L * B * T * C; // residual3
    act_sizes[15] = B * T * C; // lnf
    act_sizes[16] = B * T; // lnf_mean
    act_sizes[17] = B * T; // lnf_rstd
    act_sizes[18] = B * T; // losses
    act_sizes[19] = L * B * T * 3*C; // qkvr
    act_sizes[20] = B * T * max(3*C, max(NH*T, Vp)); // output / scratch
}

// Backward pass is conceptually quite different from forward, because we can discard
// the activations of a layer as soon as we're done with it. This lets us aggressively
// reuse memory, so that we need far fewer tensors for backward state.
#ifdef ENABLE_CUDNN
#define NUM_BACKWARD_TENSORS 2
#else
#define NUM_BACKWARD_TENSORS 3
#endif

typedef struct {
    floatX* bt4c; // (B, T, 4*C)
    floatX* residual3; // (B, T, C)
    #ifndef ENABLE_CUDNN
    floatX* preatt; // (B, NH, T, T)
    #endif
} GradActTensors;

void fill_in_grad_act_sizes(size_t* act_sizes, size_t B, size_t T, GPT2Config config) {
    size_t C = config.channels;
    act_sizes[0] = B * T * 4 * C; // bt4c
    act_sizes[1] = B * T * C; // residual3

    #ifndef ENABLE_CUDNN
    size_t NH = config.num_heads;
    act_sizes[2] = B * NH * T * T; // preatt
    #endif
}

void* malloc_and_point(floatX** targets[], const size_t* act_sizes, size_t n) {
    size_t num_activations = 0;
    for (size_t i = 0; i < n; i++) {
        num_activations += act_sizes[i];
    }
    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, num_activations * sizeof(floatX)));
    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < n; i++) {
        *(targets[i]) = (floatX*)acts_memory_iterator;
        acts_memory_iterator += act_sizes[i] * sizeof(floatX);
    }
    return acts_memory;
}

void* malloc_and_point_activations(ActivationTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->encoded, &acts->ln1, &acts->ln1_mean, &acts->ln1_rstd, &acts->atty,
        &acts->att, &acts->attproj, &acts->residual2, &acts->ln2, &acts->ln2_mean,
        &acts->ln2_rstd, &acts->fch, &acts->fch_gelu, &acts->fcproj, &acts->residual3, &acts->lnf,
        &acts->lnf_mean, &acts->lnf_rstd, &acts->losses, &acts->qkvr, &acts->output
    };
    return malloc_and_point(ptrs, act_sizes, NUM_ACTIVATION_TENSORS);
}

void* malloc_and_point_backward(GradActTensors* acts, const size_t* act_sizes) {
    floatX** ptrs[] = {
        &acts->bt4c, &acts->residual3,
        #ifndef ENABLE_CUDNN
        &acts->preatt,
        #endif
    };
    return malloc_and_point(ptrs, act_sizes, NUM_BACKWARD_TENSORS);
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    float* master_weights;     // is NULL unless fp32 weights is enabled.
    // the activations of the model, and their sizes
    ActivationTensors acts;
    size_t act_sizes[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    size_t num_activations;
    // gradients of the activations
    GradActTensors grads_acts;
    size_t num_grad_acts;
    void* grads_acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after a forward pass with targets, will be populated with the mean loss
    float accumulated_mean_loss; // Mean loss after aggregating it on all GPUs
    floatX* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    float* cpu_losses_fp32; // same but fp32
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
    int use_master_weights;
    int recompute;
    // todo - if other functions need cpu scratch buffers in the future, reuse as generic scratch?
    int* workload_indices; // encoder_backward, B*T*num_c_groups (int)
    int4* bucket_info;     // encoder_backward, B*T*num_c_groups (int4) - size for worst case
} GPT2;

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path) {

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = model_header[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }
    if (PRECISION_MODE == PRECISION_BF16 && version != 5) {
        fprintf(stderr, "Precision is configured as BF16 but model at %s is not.\n", checkpoint_path);
        fprintf(stderr, "---> HINT: are you sure you're loading a _bf16.bin file?\n");
        exit(EXIT_FAILURE);
    }
    if (PRECISION_MODE == PRECISION_FP32 && version != 3) {
        fprintf(stderr, "Precision is configured as FP32 but model at %s is not.\n", checkpoint_path);
        fprintf(stderr, "---> HINT: to turn on FP32 you have to compile like: `make train_gpt2cu PRECISION=FP32`\n");
        fprintf(stderr, "---> HINT: are you sure you're loading a .bin file without any _bf16 in the name?\n");
        exit(EXIT_FAILURE);
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate space for all the parameters and read them in
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);

    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }

    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);

    // read in all the parameters from file and copy them to device
    float* params_memory_cpu = (float*)mallocCheck(model->num_parameters_bytes);
    freadCheck(params_memory_cpu, 1, model->num_parameters_bytes, model_file);
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);
    fcloseCheck(model_file);

    // other inits
    model->acts_memory = NULL;
    model->grads_memory = NULL;
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    model->grads_acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->cpu_losses = NULL;
    model->cpu_losses_fp32 = NULL;
    model->workload_indices = NULL;
    model->bucket_info = NULL;
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f will designate no loss
    model->rng_state = 13371337;
    model->use_master_weights = 1; // keep master weights copy in float for optim update?
    model->recompute = 1; // default to recompute gelu during backward
}

void gpt2_build_from_random(GPT2 *model, int depth) {
    // init random (training from scratch)

    // parameterize the size of gpt2 based only on the depth of the model (num_layers)
    model->config.num_layers = depth;
    // follows GPT-2 sizes
    int channels, num_heads;
    if      (depth == 6)  { channels = 384; num_heads = 6; } // gpt2-tiny (30M)
    else if (depth == 12) { channels = 768; num_heads = 12; } // gpt2 (124M)
    else if (depth == 24) { channels = 1024; num_heads = 16; } // gpt2-medium (350M)
    else if (depth == 36) { channels = 1280; num_heads = 20; } // gpt2-large (774M)
    else if (depth == 48) { channels = 1600; num_heads = 25; } // gpt2-xl (1558M)
    else { fprintf(stderr, "Unsupported depth for now\n"); exit(EXIT_FAILURE); }
    model->config.channels = channels;
    model->config.num_heads = num_heads;
    model->config.max_seq_len = 1024;
    model->config.vocab_size = 50257;
    model->config.padded_vocab_size = 50304; // padded to 128

    // fill in all the parameter tensor dimensions and types
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }
    // create memory for model parameters on the device
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);

    // allocate and random init the memory for all the parameters with GPT-2 schema
    // weights ~N(0, 0.02), biases 0, c_proj weights ~N(0, 0.02/(2*L)**0.5)
    // NOTE: assuming all parameters are of the type floatX, could be relaxed later
    mt19937_state init_rng;
    manual_seed(&init_rng, 42);
    floatX* params_memory_cpu = (floatX*)mallocCheck(model->num_parameters_bytes);
    memset(params_memory_cpu, 0, model->num_parameters_bytes);
    // fill in all the weights with random values
    float residual_scale = 1.0f / sqrtf(2.0f * model->config.num_layers);
    // we have to init all these tensors exactly in the order that PyTorch initializes them
    // so that we can match them up and get correctness and exactly the same initial conditions
    size_t L = model->config.num_layers;
    size_t offset = 0;
    for (int l = 0; l < L; l++) {
        offset = 0;
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            // the layernorm parameters are all initialized to 1
            if (l == 0 && (i == 2 || i == 8 || i == 14)) { // only at l = 0 to init these just once
                for (size_t j = 0; j < model->param_elements[i]; j++) {
                    params_memory_cpu[offset + j] = 1.0f;
                }
            }
            // weights tensors are handled here
            if ((l == 0 && (i == 0 || i == 1)) // only at l = 0, init the wte and wpe tensors
              || i == 4 || i == 6 || i == 10 || i == 12) {
                int n = model->param_elements[i];
                size_t layer_offset = 0;
                if (i == 0) {
                    // for wte tensor (padded vocab) override to init V instead of Vp rows
                    n = model->config.vocab_size * model->config.channels;
                }
                if (i == 4 || i == 6 || i == 10 || i == 12) {
                    // weight tensors, we are only initializing layer l
                    assert(n % L == 0);
                    n = n / L;
                    layer_offset = l * n;
                }
                // in GPT-2, the projections back into the residual stream are additionally
                // scaled by 1/sqrt(2*L) for training stability
                float scale = (i == 6 || i == 12) ? 0.02f * residual_scale : 0.02f;
                // okay let's draw the random numbers and write them
                float *fp32_buffer = (float*)mallocCheck(n * sizeof(float));
                normal_(fp32_buffer, n, 0.0f, scale, &init_rng);
                for (size_t j = 0; j < n; j++) {
                    params_memory_cpu[offset + layer_offset + j] = (floatX)fp32_buffer[j];
                }
                free(fp32_buffer);
            }
            offset += model->param_elements[i];
        }
    }

    // copy them to GPU
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);

    // other inits and defaults
    model->acts_memory = NULL;
    model->grads_memory = NULL;
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    model->grads_acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->cpu_losses = NULL;
    model->cpu_losses_fp32 = NULL;
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f designates no loss
    model->rng_state = 13371337;
    model->use_master_weights = 1; // keep master weights copy in float for optim update?
    model->recompute = 1; // default to recompute gelu during backward
}

void gpt2_forward(GPT2 *model, int* inputs, int* targets, size_t B, size_t T, int grad_accum_steps=1) {
    NVTX_RANGE_FN();
    // targets are optional and could be NULL
    // in this function we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    size_t V = model->config.vocab_size;
    size_t Vp = model->config.padded_vocab_size;
    size_t L = model->config.num_layers;
    size_t NH = model->config.num_heads;
    size_t C = model->config.channels;

    // validate inputs, all indices must be in the range [0, V)
    for(int i = 0; i < B * T; i++) {
        assert(0 <= inputs[i] && inputs[i] < V);
        if (targets != NULL) {
            assert(0 <= targets[i] && targets[i] < V);
        }
    }

    // allocate space for all the activations if needed (done here, lazily)
    if(model->acts_memory == NULL) {
        // record the current B,T as well
        model->batch_size = B;
        model->seq_len = T;
        // allocate the space
        fill_in_activation_sizes(model->act_sizes, B, T, model->config, model->recompute);
        size_t num_activations = 0;
        for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
            num_activations += model->act_sizes[i];
        }
        model->num_activations = num_activations;
        printf0("allocating %d MiB for activations\n", (int)round(num_activations * sizeof(floatX) / (1024 * 1024)));
        model->acts_memory = malloc_and_point_activations(&model->acts, model->act_sizes);
        // also create memory for caching inputs and targets
        cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
        cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(floatX)));
        cudaCheck(hipHostMalloc((void**)&model->cpu_losses_fp32, B * T * sizeof(float)));
    } else {
        // validate B,T is consistent with how we've allocated the memory before
        // in principle we could get more clever here in the future, for now this is safest
        if (B != model->batch_size || T != model->seq_len) {
            printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
            exit(EXIT_FAILURE);
        }
    }

    // copy inputs/targets to the model
    // todo - inputs is copied on default stream so this synchronises CPU/GPU for now
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    if (targets != NULL) {
        cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    }

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C); // encoding goes into residual[0]

    // first layernorm isn't fused
    layernorm_forward(acts.ln1, acts.ln1_mean, acts.ln1_rstd, acts.encoded, params.ln1w, params.ln1b, B, T, C);

    for (int l = 0; l < L; l++) {
        NvtxRange layer_range("Layer", l);

        floatX* residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = acts.ln1 + l * B * T * C;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_attproj = acts.attproj + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = acts.ln2 + l * B * T * C;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        // reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
        // very useful because we dramatically reduce VRAM usage, and may be able to fit larger batch size
        floatX* l_fch_gelu = (model->recompute == 0) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        floatX* l_fcproj = acts.fcproj + l * B * T * C;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;

        // now do the forward pass
        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        floatX* scratch = (floatX*)acts.output;
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH);
        #endif

        matmul_forward_cublaslt(l_attproj, l_atty, l_attprojw, l_attprojb, B, T, C, C);
        fused_residual_forward5(l_residual2, l_ln2, l_ln2_mean, l_ln2_rstd, residual, l_attproj, l_ln2w, l_ln2b, B*T, C);
        matmul_forward_cublaslt(l_fch, l_ln2, l_fcw, l_fcb, B, T, C, 4*C);
        gelu_forward(l_fch_gelu, l_fch, B*T*4*C);
        matmul_forward_cublaslt(l_fcproj, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C);

        // OK, fusion across blocks.
        if(l+1 != L) {
            floatX* l_ln1 = acts.ln1 + (l + 1) * B * T * C;
            floatX* l_ln1_mean = acts.ln1_mean + (l + 1) * B * T;
            floatX* l_ln1_rstd = acts.ln1_rstd + (l + 1) * B * T;
            const floatX* l_ln1w = params.ln1w + (l + 1) * C;
            const floatX* l_ln1b = params.ln1b + (l + 1) * C;
            fused_residual_forward5(l_residual3, l_ln1, l_ln1_mean, l_ln1_rstd, l_residual2, l_fcproj, l_ln1w, l_ln1b,
                                    B * T, C);
        } else {
            fused_residual_forward5(l_residual3, acts.lnf, acts.lnf_mean, acts.lnf_rstd, l_residual2, l_fcproj,
                                    params.lnfw, params.lnfb,
                                    B * T, C);
        }
    }

    matmul_forward_cublaslt(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp);

    // also forward the cross-entropy loss function if we have the targets
    if (targets != NULL) {
        NvtxRange classifier_and_loss_range("classifier_and_loss");
        // fused classifier: does the forward pass and first part of the backward pass
        const float dloss = 1.0f / (B * T * grad_accum_steps); // results in the uniform average loss over all elements
        fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp);
        // for convenience also evaluate the mean loss (TODO re-think this compute+sync point)
        cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(floatX), hipMemcpyDeviceToHost));
        float mean_loss = 0.0f;
        for (int i = 0; i < B*T; i++) {
            float loss = (float)(model->cpu_losses[i]);
            model->cpu_losses_fp32[i] = loss;
            mean_loss += loss;
        }
        mean_loss /= B*T*grad_accum_steps;
        model->mean_loss = mean_loss;
    } else {
        // if we don't have targets, we don't have loss
        model->mean_loss = -1.0f;
    }
}

void gpt2_zero_grad(GPT2 *model) {
    NVTX_RANGE_FN();
    if (model->grads_memory != NULL) {
        cudaCheck(hipMemset(model->grads_memory, 0, model->num_parameters * sizeof(floatX)));
    }
}

void gpt2_backward(GPT2 *model, int* inputs) {
    NVTX_RANGE_FN();
    // double check we forwarded previously, with targets
    if (model->mean_loss == -1.0f) {
        printf("Error: must forward with targets before backward\n");
        exit(EXIT_FAILURE);
    }

    // lazily allocate the memory for gradients of the weights and activations, if needed
    if (model->grads_memory == NULL) {
        // allocate buffers for weight gradients
        printf0("allocating %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
        model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof);
        // we're going to be clever for the activations backward pass. we don't need to exactly
        // mirror the forward pass activations and we will save memory.
        size_t bw_act_sizes[NUM_ACTIVATION_TENSORS];
        fill_in_grad_act_sizes(bw_act_sizes, model->batch_size, model->seq_len, model->config);
        // count up and allocate the space
        model->num_grad_acts = 0;
        for (size_t i = 0; i < NUM_BACKWARD_TENSORS; i++) {
            model->num_grad_acts += bw_act_sizes[i];
        }
        printf0("allocating %d MiB for activation gradients\n", (int)round(model->num_grad_acts * sizeof(floatX) / (1024 * 1024)));
        model->grads_acts_memory = malloc_and_point_backward(&model->grads_acts, bw_act_sizes);
        // init gradients of parameters and activations to zero
        gpt2_zero_grad(model);
        // initialise cpu scratch buffers for encoder backward
        size_t num_c_groups = model->config.channels / (WARP_SIZE * x128::size);
        assert((size_t)(model->batch_size * model->seq_len) * num_c_groups < (1ULL<<31ULL)); // todo - maybe an issue for llama3-400B(?)
        model->workload_indices = (int*)mallocCheck(sizeof(int) * model->batch_size * model->seq_len * num_c_groups);
        model->bucket_info = (int4*)mallocCheck(sizeof(int4) * model->batch_size * model->seq_len * num_c_groups);
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    size_t B = model->batch_size;
    size_t T = model->seq_len;
    size_t Vp = model->config.padded_vocab_size;
    size_t L = model->config.num_layers;
    size_t NH = model->config.num_heads;
    size_t C = model->config.channels;

    // backward pass: go in the reverse order of the forward pass, and call backward() functions
    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;
    GradActTensors grads_acts = model->grads_acts;

    // reset residual stream gradients (put here to work with gradient accumulation)
    cudaCheck(hipMemset(model->grads_acts.residual3, 0, B * T * C * sizeof(floatX)));

    // re-use the output buffer of the forward pass as a scratchpad during backward pass
    float*  scratchF = (float*)acts.output;
    floatX* scratchX = (floatX*)acts.output;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(grads_acts.bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    floatX* dresidual = (floatX*)grads_acts.residual3; // the main buffer holding the gradient in the backward pass
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, scratchF, grads_acts.bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C);

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        NvtxRange layer_range("Layer", l);

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        floatX* dl_ln1w = grads.ln1w + l * C;
        floatX* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * 3*C * C;
        floatX* dl_qkvb = grads.qkvb + l * 3*C;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = grads.attprojb + l * C;
        floatX* dl_ln2w = grads.ln2w + l * C;
        floatX* dl_ln2b = grads.ln2b + l * C;
        floatX* dl_fcw = grads.fcw + l * 4*C * C;
        floatX* dl_fcb = grads.fcb + l * 4*C;
        floatX* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        floatX* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = acts.ln1 + l * B * T * C;
        floatX* l_ln1_mean = acts.ln1_mean + l * B * T;
        floatX* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = acts.ln2 + l * B * T * C;
        floatX* l_ln2_mean = acts.ln2_mean + l * B * T;
        floatX* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = (model->recompute == 0) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        // we need a B x T x C buffer; thankfully, the forward activation for lnf isn't needed anymore,
        // so we can co-opt it here.
        floatX* dl_btc = (floatX*)acts.lnf;
        floatX* dl_bt4c = (floatX*)grads_acts.bt4c;

        // start the backward pass for this layer
        if(model->recompute >= 1) {
            // recompute >= 1 means we recompute gelu. in this case,
            // l_fch_gelu is just a buffer, so re-compute the gelu from l_fch here
            gelu_forward(l_fch_gelu, l_fch, B*T*4*C);
        }
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, scratchF, B, T, 4*C, C);
        gelu_backward(dl_bt4c, l_fch, dl_bt4c, B*T*4*C);
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, scratchF, B, T, C, 4 * C);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, scratchF, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, scratchF, B, T, C, C);

        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        attention_backward_cudnn(dl_bt4c, dl_btc, l_qkvr, l_atty, (float*)l_att, B, T, NH, C);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // we need B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch;        // this is B x T x 4C, so even larger than what we need
        floatX* dl_preatt = (floatX*)grads_acts.preatt; // dedicated scratchpad allocation
        attention_backward(dl_bt4c, buffer_b, dl_preatt, scratchX, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH);
        #endif

        // QKV parameter gradients
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, scratchF, B, T, C, 3 * C);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, scratchF, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C);
    }
    encoder_backward(grads.wte, grads.wpe, scratchX, model->workload_indices, model->bucket_info,
                     dresidual, model->inputs, inputs, B, T, C, random_u32(&model->rng_state));
}

// Compute sum of a single CPU value across all GPU processes. No-op when multi-GPU is disabled.
float multi_gpu_cpu_float_sum(float value) {
#ifdef MULTI_GPU
    // note MPI doesn't support all reduce with mean, only sum
    float result;
    mpiCheck(MPI_Allreduce(&value, &result, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD));
    return result;
#else
    return value;
#endif
}

// Averages out the loss and gradients across all GPUs. No-op when multi-GPU is disabled.
// todo - this version only works if all the parameters are the same size (floatX)
void gpt2_multi_gpu_accumulate(GPT2* model, MultiGpuConfig* multi_gpu_config) {
#ifdef MULTI_GPU
    NVTX_RANGE_FN();
    if (multi_gpu_config->num_processes == 1) { return; }
    // Average all losses.
    model->accumulated_mean_loss = multi_gpu_cpu_float_sum(model->mean_loss) / multi_gpu_config->num_processes;
    if(multi_gpu_config->zero_stage == 0) {
        //  no ZERO == standard DDP: Average all gradients.
        ncclCheck(ncclAllReduce(model->grads_memory, model->grads_memory,
                                model->num_parameters,
                                ncclFloatX, ncclAvg,
                                multi_gpu_config->nccl_comm, 0));
    } else if (multi_gpu_config->zero_stage == 1) {
        // ZERO-1: Get average gradient for local shard
        floatX* local_grads_memory = (floatX*) model->grads_memory + multi_gpu_config->shard_offset;
        ncclCheck(ncclReduceScatter(model->grads_memory, local_grads_memory,
                                    multi_gpu_config->shard_num_parameters,
                                    ncclFloatX, ncclAvg,
                                    multi_gpu_config->nccl_comm, 0));
    }
#endif
}

float gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, float grad_clip, int t, MultiGpuConfig* multi_gpu_config) {
    NVTX_RANGE_FN();
    size_t num_parameters = multi_gpu_config->shard_num_parameters;
    floatX* params_memory = (floatX*)model->params_memory + multi_gpu_config->shard_offset;
    floatX* grads_memory = (floatX*)model->grads_memory + multi_gpu_config->shard_offset;

    if (model->m_memory == NULL) {
        printf0("allocating %zu MiB for AdamW optimizer state m\n", (num_parameters * sizeof(float)) >> 20);
        printf0("allocating %zu MiB for AdamW optimizer state v\n", (num_parameters * sizeof(float)) >> 20);
        cudaCheck(hipMalloc((void**)&model->m_memory, num_parameters * sizeof(float)));
        cudaCheck(hipMalloc((void**)&model->v_memory, num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->m_memory, 0, num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, num_parameters * sizeof(float)));
        if (model->use_master_weights == 1) {
            printf0("allocating %zu MiB for master copy of params\n", (num_parameters * sizeof(float)) >> 20);
            cudaCheck(hipMalloc((void**)&model->master_weights, num_parameters * sizeof(float)));
            copy_and_cast_kernel<<<CEIL_DIV(num_parameters, 512), 512>>>(model->master_weights, params_memory, num_parameters);
            cudaCheck(hipGetLastError());
        }
    }

    // gradient clipping
    // repurposing this buffer (which isn't needed now) to write grad norm into it
    float* grad_norm_squared = (float*)model->acts.output;
    global_norm_squared(grad_norm_squared, (floatX*)model->grads_memory, model->num_parameters);
    // transfer the gradient norm to CPU
    float grad_norm_squared_cpu = 0.0f;
    cudaCheck(hipMemcpy(&grad_norm_squared_cpu, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));
    if(!isfinite(grad_norm_squared_cpu)) {
        // may happen due to some issue (e.g. overflow?)
        // TODO: later may want to keep a global counter of instabilities like this
        printf0("[WARNING]: grad norm is not finite, skipping AdamW update\n");
        return -1.0f;
    }
    float grad_norm_cpu = sqrtf(grad_norm_squared_cpu);
    float grad_scale = (grad_norm_cpu > grad_clip) ? grad_clip / grad_norm_cpu : 1.0f;

    // AdamW update
    int block_size = 512;
    int num_blocks = CEIL_DIV(num_parameters, block_size);
    unsigned int seed = random_u32(&model->rng_state);
    float learning_rate_inv_beta1_correction = learning_rate*(1.0f/1.0f - powf(beta1, t));
    float inv_beta2_correction = 1.0f/(1.0f - powf(beta2, t));
    float learning_rate_weight_decay = learning_rate * weight_decay;
    assert(num_parameters % block_size == 0); //bounds check
    adamw_kernel3<<<num_blocks, block_size>>>(params_memory, model->master_weights, grads_memory,
                                              model->m_memory, model->v_memory, beta1, beta2, learning_rate_inv_beta1_correction,
                                              inv_beta2_correction, eps, learning_rate_weight_decay, grad_scale, seed);
    cudaCheck(hipGetLastError());
    return grad_norm_cpu;
}

void gpt2_multi_gpu_gather(GPT2 *model, MultiGpuConfig* multi_gpu_config)
{
#ifdef MULTI_GPU
    if (multi_gpu_config->num_processes == 1) { return; } // 1 process => noop
    if (multi_gpu_config->zero_stage == 1) {
        // gather updated shards of model->params_memory from each process
        ncclCheck(ncclAllGather((floatX*)model->params_memory + multi_gpu_config->shard_offset, (floatX*)model->params_memory,
                                multi_gpu_config->shard_num_parameters, ncclFloatX,
                                multi_gpu_config->nccl_comm, 0));
    }
    cudaCheck(hipGetLastError());
#endif
}

void gpt2_free(GPT2 *model) {
    cudaCheck(hipFree(model->params_memory));
    cudaCheck(hipFree(model->grads_memory));
    cudaCheck(hipFree(model->m_memory));
    cudaCheck(hipFree(model->v_memory));
    cudaCheck(hipFree(model->master_weights));
    cudaCheck(hipFree(model->acts_memory));
    cudaCheck(hipFree(model->grads_acts_memory));
    cudaCheck(hipFree(model->inputs));
    cudaCheck(hipFree(model->targets));
    hipHostFree(model->cpu_losses);
    hipHostFree(model->cpu_losses_fp32);
    free(model->workload_indices);
    free(model->bucket_info);
}

// ----------------------------------------------------------------------------
// common init & free code for train/test/profile
void common_start(bool override_enable_tf32 = true, bool print_device_info = true) {
    hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx);
    if (print_device_info) {
        printf("[System]\n");
        printf("Device %d: %s\n", multi_gpu_config.local_device_idx, deviceProp.name);
    }

    // set up cuBLAS and cuBLASLt (and cuDNN if enabled)
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;
    cublasCheck(hipblasSetMathMode(cublas_handle, enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH));
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;

    create_cudnn();
}

void common_free(GPT2 &model) {
    gpt2_free(&model);
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    destroy_cudnn();
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip everything below this point
// ----------------------------------------------------------------------------
// sampler: takes probabilities and samples integers from them

int sample_softmax(const float* logits, int n, float coin) {
    // sample index from logits (converted to probabilities using softmax)
    // coin is a random number in [0, 1), usually from random_f32()
    double norm = 0;
    for (int i = 0; i < n; i++) {
        norm += expf(logits[i]);
    }
    // instead of dividing all exp(logits), we can just multiply coin.
    coin *= norm;
    float cdf = 0.0f;
    for (int i = 0; i < n; i++) {
        cdf += expf(logits[i]);
        if (coin < cdf) {
            return i;
        }
    }
    return n - 1; // in case of rounding errors
}

// ----------------------------------------------------------------------------
// Logger lite, will probably grow/change some over time

typedef struct {
    FILE *logfile;
    int flush_every; // every how many steps to flush the log
} Logger;

void logger_init(Logger *logger, const char *filename) {
    logger->flush_every = 10;
    logger->logfile = NULL;
    // only rank 0 process will log
    if (filename != NULL && multi_gpu_config.process_rank == 0) {
        logger->logfile = fopenCheck(filename, "w");
    }
}

void logger_log_eval(Logger *logger, int step, float val) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d eval:%.4f\n", step, val);
    }
}

void logger_log_val(Logger *logger, int step, float val_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d tel:%.4f\n", step, val_loss);
    }
}

void logger_log_train(Logger *logger, int step, float train_loss) {
    if (logger->logfile != NULL) {
        fprintf(logger->logfile, "s:%d trl:%.4f\n", step, train_loss);
        if (step % logger->flush_every == 0) { fflush(logger->logfile); }
    }
}

void logger_free(Logger *logger) {
    if (logger->logfile != NULL) { fclose(logger->logfile); }
}

// ----------------------------------------------------------------------------
// CLI, poor man's argparse

void error_usage() {
    fprintf(stderr, "Usage:   ./train_gpt2cu [options]\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -i <string> train data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_train.bin)\n");
    fprintf(stderr, "  -j <string> val data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_val.bin)\n");
    fprintf(stderr, "  -e <string> input from model at this filename (default = gpt2_124M_bf16.bin)\n");
    fprintf(stderr, "  -o <string> output log file (default = NULL)\n");
    fprintf(stderr, "  -b <int>    (per-GPU, micro) batch size B (default = 4)\n");
    fprintf(stderr, "  -t <int>    sequence length T (default = 1024)\n");
    fprintf(stderr, "  -d <int>    total desired batch size (default = B * T * num_processes, i.e. no grad accumulation\n");
    fprintf(stderr, "  -l <float>  learning rate (default = 3e-4f)\n");
    fprintf(stderr, "  -u <int>    learning rate warmup iterations (default = 0, no warmup)\n");
    fprintf(stderr, "  -q <float>  learning rate decay: final fraction, at end of training (default = 1.0 (no decay))\n");
    fprintf(stderr, "  -c <float>  weight decay (default = 0.0f)\n");
    fprintf(stderr, "  -x <int>    max_steps of optimization to run (-1 (default) = disable, run 1 epoch)\n");
    fprintf(stderr, "  -v <int>    val_loss_every, how often we evaluate val loss (default = 20)\n");
    fprintf(stderr, "  -m <int>    val_max_batches, up to how many val batches to estimate val loss? (default = 20)\n");
    fprintf(stderr, "  -s <int>    sample_every, how often we inference the model (default = 20)\n");
    fprintf(stderr, "  -g <int>    genT, how many steps of inference we do (default = 64)\n");
    fprintf(stderr, "  -a <int>    overfit a single batch? 0/1. useful for debugging\n");
    fprintf(stderr, "  -f <int>    enable_tf32 override (default: 1, set to 0 to disable tf32)\n");
    fprintf(stderr, "  -w <int>    keep f32 copy of weights for the optimizer? (default: 1)\n");
    fprintf(stderr, "  -z <int>    zero_stage, Zero Optimization Stage, 0,1,2,3 (default = 0)\n");
    fprintf(stderr, "  -r <int>    recompute: saves memory at cost of speed. (default = 1), 0 = none. 1 = recompute gelu\n");
    fprintf(stderr, "  -h <int>    hellaswag eval run? (default = 0)\n");
    exit(EXIT_FAILURE);
}

// ----------------------------------------------------------------------------
// main training loop
int main(int argc, char *argv[]) {
    multi_gpu_config = multi_gpu_config_init(&argc, &argv);

    // read in the (optional) command line arguments
    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* load_filename = "gpt2_124M_bf16.bin"; // bf16 weights of the model
    const char* output_log_file = NULL;
    int B = 4; // batch size
    int T = 1024; // sequence length max
    int total_batch_size = -1; // will be calculated down below later, if not provided
    float learning_rate = 3e-4f;
    int warmup_iterations = 0;
    float final_learning_rate_frac = 1.0f; // final fraction of learning rate, at end of training
    float weight_decay = 0.0f;
    int val_loss_every = 20; // every how many steps do we eval validation loss?
    int val_max_batches = 20; // how many batches max do we eval for validation loss?
    int sample_every = 20; // every how many steps to do inference?
    int genT = 64; // number of steps of inference we will do
    int overfit_single_batch = 0; // useful for debugging, 1 = only load a single data batch once
    int max_steps = -1;
    int override_enable_tf32 = 1;
    int use_master_weights = 1;
    int recompute = 1; // recompute during backward setting, 0 = none, 1 = recompute gelu
    int zero_stage = 0; // Zero Optimization Stage for Multi-GPU training
    float grad_clip  = 1.0f;
    int hellaswag_eval = 0;
    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } // must have arg after flag
        if (argv[i][0] != '-') { error_usage(); } // must start with dash
        if (strlen(argv[i]) != 2) { error_usage(); } // must be -x (one dash, one letter)
        // read in the args
        if (argv[i][1] == 'i') { train_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'j') { val_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'e') { load_filename = argv[i+1]; }
        else if (argv[i][1] == 'o') { output_log_file = argv[i+1]; }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } // Per-GPU (micro) batch size
        else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'd') { total_batch_size = atoi(argv[i+1]); }
        else if (argv[i][1] == 'l') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'u') { warmup_iterations = atoi(argv[i+1]); }
        else if (argv[i][1] == 'q') { final_learning_rate_frac = atof(argv[i+1]); }
        else if (argv[i][1] == 'c') { weight_decay = atof(argv[i+1]); }
        else if (argv[i][1] == 'x') { max_steps = atoi(argv[i+1]); }
        else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_batches = atoi(argv[i+1]); }
        else if (argv[i][1] == 's') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else if (argv[i][1] == 'a') { overfit_single_batch = atoi(argv[i+1]); }
        else if (argv[i][1] == 'f') { override_enable_tf32 = atoi(argv[i+1]); }
        else if (argv[i][1] == 'w') { use_master_weights = atoi(argv[i+1]); }
        else if (argv[i][1] == 'c') { grad_clip = atof(argv[i+1]); }
        else if (argv[i][1] == 'z') { zero_stage = atoi(argv[i+1]); }
        else if (argv[i][1] == 'r') { recompute = atoi(argv[i+1]); }
        else if (argv[i][1] == 'h') { hellaswag_eval = atoi(argv[i+1]); }
        else { error_usage(); }
    }
    // should do a bit more error checking here
    assert(warmup_iterations >= 0);
    // calculate a sensible default for total batch size by assuming no gradient accumulation
    if (total_batch_size == -1) { total_batch_size = B * T * multi_gpu_config.num_processes; }
    // if we're only overfitting a single batch for debugging, let's overfit the first batch
    // from val instead of train split, because val is smaller and faster. (train_gpt2.py does the same)
    if (overfit_single_batch == 1) { train_data_pattern = val_data_pattern; }
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| Parameter             | Value                                              |\n");
    printf0("+-----------------------+----------------------------------------------------+\n");
    printf0("| train data pattern    | %-50s |\n", train_data_pattern);
    printf0("| val data pattern      | %-50s |\n", val_data_pattern);
    printf0("| output log file       | %-50s |\n", output_log_file == NULL ? "NULL" : output_log_file);
    printf0("| micro batch size B    | %-50d |\n", B);
    printf0("| sequence length T     | %-50d |\n", T);
    printf0("| total batch size      | %-50d |\n", total_batch_size);
    printf0("| learning rate (LR)    | %-50e |\n", learning_rate);
    printf0("| warmup iterations     | %-50d |\n", warmup_iterations);
    printf0("| final LR fraction     | %-50e |\n", final_learning_rate_frac);
    printf0("| weight decay          | %-50e |\n", weight_decay);
    printf0("| grad_clip             | %-50e |\n", grad_clip);
    printf0("| max_steps             | %-50d |\n", max_steps);
    printf0("| val_loss_every        | %-50d |\n", val_loss_every);
    printf0("| val_max_batches       | %-50d |\n", val_max_batches);
    printf0("| sample_every          | %-50d |\n", sample_every);
    printf0("| genT                  | %-50d |\n", genT);
    printf0("| overfit_single_batch  | %-50d |\n", overfit_single_batch);
    printf0("| use_master_weights    | %-50s |\n", use_master_weights ? "enabled" : "disabled");
    printf0("| recompute             | %-50d |\n", recompute);
    printf0("+-----------------------+----------------------------------------------------+\n");

    common_start(override_enable_tf32, false); // common init code for train/test/profile

    const char* precision_str = (PRECISION_MODE == PRECISION_FP32)
                              ? (cublas_compute == HIPBLAS_COMPUTE_32F_FAST_TF32 ? "TF32" : "FP32")
                              : (PRECISION_MODE == PRECISION_FP16 ? "FP16" : "BF16");

    printf0("| device                | %-50s |\n", deviceProp.name);
    printf0("| precision             | %-50s |\n", precision_str);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build the GPT-2 model
    GPT2 model;
    // if load_filename is of the form "dX" where X is an integer (e.g. d12), then we build
    // a random model with the depth of the model specified by X (e.g. 12). otherwise interpret
    // this variable as a checkpoint filename, and load that checkpoint
    assert(strlen(load_filename) >= 2);
    if (load_filename[0] == 'd') {
        int depth = atoi(load_filename + 1);
        if (depth > 1 && depth <= 1000) { // we're not going to train models this big right? heh
            gpt2_build_from_random(&model, depth);
        } else {
            exit(EXIT_FAILURE);
        }
    } else {
        gpt2_build_from_checkpoint(&model, load_filename);
    }

    model.use_master_weights = use_master_weights;
    model.recompute = recompute;
    printf0("| load_filename         | %-50s |\n", load_filename);
    printf0("| max_sequence_length T | %-50d |\n", model.config.max_seq_len);
    printf0("| vocab_size V          | %-50d |\n", model.config.vocab_size);
    printf0("| padded_vocab_size Vp  | %-50d |\n", model.config.padded_vocab_size);
    printf0("| num_layers L          | %-50d |\n", model.config.num_layers);
    printf0("| num_heads NH          | %-50d |\n", model.config.num_heads);
    printf0("| channels C            | %-50d |\n", model.config.channels);
    printf0("| num_parameters        | %-50zu |\n", model.num_parameters);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build DataLoaders for both train and val
    DataLoader train_loader, val_loader;
    dataloader_init(&train_loader, train_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    dataloader_init(&val_loader, val_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    int train_num_batches = (max_steps == -1) ? train_loader.num_batches : max_steps; // default = 1 epoch
    int val_num_batches = train_loader.num_batches < val_max_batches ? train_loader.num_batches : val_max_batches;
    printf0("| train_num_batches     | %-50d |\n", train_num_batches);
    printf0("| val_num_batches       | %-50d |\n", val_num_batches);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // build an EvalLoader for HellaSwag
    EvalLoader eval_loader;
    const char* hellaswag_path = "dev/data/hellaswag/hellaswag_val.bin";
    const char hellaswag_available = access(hellaswag_path, F_OK) == 0;
    const char run_hellaswag = hellaswag_eval && hellaswag_available;
    if (run_hellaswag) {
        evalloader_init(&eval_loader, hellaswag_path, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes);
    }
    printf0("| run hellaswag         | %-50s |\n", run_hellaswag ? "yes" : "no");
    printf0("+-----------------------+----------------------------------------------------+\n");

    // pretty print in a table the multi-gpu configuration as well
    set_zero_configs(&multi_gpu_config, zero_stage, model.num_parameters);
    printf0("| num_processes         | %-50d |\n", multi_gpu_config.num_processes);
    printf0("| zero_stage            | %-50d |\n", multi_gpu_config.zero_stage);
    printf0("+-----------------------+----------------------------------------------------+\n");

    // prints outside of pretty table to here and below
    if (!hellaswag_available) {
        printf0("HellaSwag eval not found at %s, skipping its evaluation\n", hellaswag_path);
        printf0("You can run `python dev/data/hellaswag.py` to export and use it with `-h 1`.\n");
    }
    // more prints related to allocations from gpt2_build_from_checkpoint down here to not mess up our table above
    printf0("num_parameters: %zu => bytes: %zu\n", model.num_parameters, model.num_parameters_bytes);
    printf0("allocated %d MiB for model parameters\n", (int)round(model.num_parameters_bytes / (1024 * 1024)));

    // figure out gradient accumulation from the desired total batch size
    int tokens_per_fwdbwd = B * T * multi_gpu_config.num_processes; // one micro-batch processes this many tokens
    assert(total_batch_size % tokens_per_fwdbwd == 0);
    int grad_accum_steps = total_batch_size / tokens_per_fwdbwd;
    printf0("batch_size B=%d * seq_len T=%d * num_processes=%d and total_batch_size=%d\n",
            B, T, multi_gpu_config.num_processes, total_batch_size);
    printf0("=> setting grad_accum_steps=%d\n", grad_accum_steps);

    // set up the Logger
    Logger logger;
    logger_init(&logger, output_log_file);

    // set up the Tokenizer
    Tokenizer tokenizer;
    tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");

    // some memory for generating samples from the model
    unsigned long long rng_state = 1337;
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float*  cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    // train
    hipEvent_t start, end;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&end));
    cudaCheck(hipProfilerStart());
    double total_sum_iteration_time_s = 0.0;
    float ema_tokens_per_second = 0.0f;
    for (int step = 0; step <= train_num_batches; step++) {
        NvtxRange step_range("Train step", step);

        int last_step = step == train_num_batches;

        // once in a while estimate the validation loss
        if (step % val_loss_every == 0 || last_step) {
            NvtxRange validation_range("validation");
            float val_loss = 0.0f;
            dataloader_reset(&val_loader);
            for (int i = 0; i < val_num_batches; i++) {
                dataloader_next_batch(&val_loader);
                gpt2_forward(&model, val_loader.inputs, val_loader.targets, B, T);
                val_loss += model.mean_loss;
            }
            val_loss /= val_num_batches;
            val_loss = multi_gpu_cpu_float_sum(val_loss) / multi_gpu_config.num_processes;
            printf0("val loss %f\n", val_loss);
            logger_log_val(&logger, step, val_loss);
        }

        // once in a while estimate HellaSwag accuracy
        if (run_hellaswag &&
           ((step > 0 && step % val_loss_every == 0) || last_step)) {
            NvtxRange evaluation_range("evaluation");
            float eval_acc_norm = 0.0f;
            evalloader_reset(&eval_loader);
            for (int i = 0; i < eval_loader.num_batches; i++) {
                if (i % 10 == 0) { printf("evaluating HellaSwag: %d/%d\r", i, eval_loader.num_batches); }
                evalloader_next_batch(&eval_loader);
                gpt2_forward(&model, eval_loader.inputs, eval_loader.targets, B, T);
                int correct = evalloader_stat_losses(&eval_loader, model.cpu_losses_fp32);
                eval_acc_norm += (float)correct;
            }
            // careful because not all ranks may have the exact same allocation of number of examples
            eval_acc_norm = multi_gpu_cpu_float_sum(eval_acc_norm);
            printf0("HellaSwag: %d/%d = %f\n", (int)eval_acc_norm, eval_loader.num_examples, eval_acc_norm / eval_loader.num_examples);
            logger_log_eval(&logger, step, eval_acc_norm / eval_loader.num_examples);
        }

        // once in a while do model inference to print generated text
        if (multi_gpu_config.process_rank == 0 && (step > 0 && (step % sample_every) == 0 || last_step)) {
            NvtxRange generation_range("generation");
            // fill up gen_tokens with the <|endoftext|> token, which kicks off the generation
            int eot_token = tokenizer.eot_token;
            for(int i = 0; i < B * T; ++i) {
                gen_tokens[i] = eot_token;
            }
            // now sample from the model autoregressively
            printf("generating:\n---\n");
            for (int t = 1; t < genT; t++) {
                NvtxRange generation_range("Generation step", t);
                // note that inference is very wasteful here because for each token
                // we re-calculate the forward pass for all of (B,T) positions from scratch
                // but the inference here is just for sanity checking anyway
                // and we can maybe optimize a bit more later, with careful tests
                gpt2_forward(&model, gen_tokens, NULL, B, T);
                // furthermore, below we're only using b=0 (i.e. the first row) of all B rows
                // we're in principle running B "inference streams" in parallel here
                // only using position 0 because it's a bit faster (copy less probs from GPU -> CPU)
                // get the V-dimensional vector probs[0, t-1, :]
                floatX* logits = model.acts.output + (t - 1) * model.config.padded_vocab_size;
                // move probs back to CPU and sample (note we only move the first vocab_size logits, ignoring the padding)
                cudaCheck(hipMemcpy(cpu_logits_raw, logits, model.config.vocab_size * sizeof(floatX), hipMemcpyDeviceToHost));
                // convert to FP32 into cpu_logits (this does nothing useful if floatX == float)
                for (int i = 0; i < model.config.vocab_size; i++) {
                    cpu_logits[i] = (float)cpu_logits_raw[i];
                }

                float coin = random_f32(&rng_state);
                int next_token = sample_softmax(cpu_logits, model.config.vocab_size, coin);
                gen_tokens[t] = next_token;
                // print the generated token, either using the Tokenizer or a fallback
                if (tokenizer.init_ok) {
                    const char* token_str = tokenizer_decode(&tokenizer, next_token);
                    safe_printf(token_str);
                } else {
                    // fall back to printing the token id
                    printf("%d ", next_token);
                }
                fflush(stdout);
            }
            printf("\n---\n");
        }

        // bit confusing: we want to make sure to eval and sample on 0th iteration
        // but also after the very last iteration. so we loop for step <= train_num_batches
        // instead of just < train_num_batches (one extra due to <=), only to do
        // the validation/sampling one last time, and then we break right here as we're done.
        if (last_step) { break; }

        // --------------- TRAINING SECTION BEGIN -----------------
        // do one training step, doing forward/backward/update on total_batch_size tokens
        hipEventRecord(start);
        // gradient accumulation loop over micro-batches
        float lossf = 0.0f; // for getting the mean loss over the accumulation steps
        for (int micro_step = 0; micro_step < grad_accum_steps; micro_step++) {
            // fetch the next data batch
            // and if we're overfitting a single batch, we'll only call this a single time
            if (overfit_single_batch == 0 ||
               (overfit_single_batch == 1 && step == 0 && micro_step == 0)) {
                dataloader_next_batch(&train_loader);
            }
            // forward pass. note that we pass in grad_accum_steps, which scales down the loss
            gpt2_forward(&model, train_loader.inputs, train_loader.targets, B, T, grad_accum_steps);
            lossf += model.mean_loss; // the mean_loss was normalized by grad_accum_steps inside gpt2_forward
            // backward pass. all model params accumulate gradients with += inside this inner loop
            gpt2_backward(&model, train_loader.inputs);
        }
        // override the mean loss, accounting for the gradient accumulation loop
        // this is esp important to do here in multigpu update below, where model.mean_loss gets allreduced
        model.mean_loss = lossf;
        // update the parameters
        gpt2_multi_gpu_accumulate(&model, &multi_gpu_config);
        // learning rate schedule: warmup linearly to max LR, then cosine decay to LR * final_learning_rate_frac
        float step_learning_rate = learning_rate;
        if (step < warmup_iterations) {
            step_learning_rate = learning_rate * ((float)(step + 1)) / warmup_iterations;
        } else {
            float decay_ratio = ((float)(step - warmup_iterations)) / (train_num_batches - warmup_iterations);
            assert(0.0f <= decay_ratio && decay_ratio <= 1.0f);
            float coeff = 0.5f * (1.0f + cosf(M_PI * decay_ratio)); // coeff starts at 1 and goes to 0
            assert(0.0f <= coeff && coeff <= 1.0f);
            float min_lr = learning_rate * final_learning_rate_frac;
            step_learning_rate = min_lr + coeff * (learning_rate - min_lr);
        }
        // update the model parameters
        float grad_norm = gpt2_update(&model, step_learning_rate, 0.9f, 0.95f, 1e-8f, weight_decay, grad_clip, step+1, &multi_gpu_config);
        gpt2_multi_gpu_gather(&model, &multi_gpu_config);
        // zero out the gradients for the next iteration
        gpt2_zero_grad(&model);
        hipEventRecord(end);
        cudaCheck(hipEventSynchronize(end)); // wait for the end event to finish to get correct timings
        // --------------- TRAINING SECTION END -------------------
        // everything that follows now is just diagnostics, prints, logging, etc.

        // todo - move or double-buffer all of this timing logic to avoid idling the GPU at this point!
        float time_elapsed_ms;
        cudaCheck(hipEventElapsedTime(&time_elapsed_ms, start, end));
        size_t tokens_processed = (size_t)multi_gpu_config.num_processes * B * T * grad_accum_steps;
        float tokens_per_second = tokens_processed / time_elapsed_ms * 1000.0f;
        float bias_corrected_ema_tokens_per_second = tokens_per_second; // by default set to non-ema version
        if (step > 0) { // consider the first batch to be a warmup (e.g. cuBLAS/cuDNN initialisation)
            total_sum_iteration_time_s += time_elapsed_ms / 1000.0f;
            // smooth out the tok/s with an exponential moving average, and bias correct just like in AdamW
            ema_tokens_per_second = 0.95f * ema_tokens_per_second + 0.05f * tokens_per_second;
            bias_corrected_ema_tokens_per_second = ema_tokens_per_second / (1.0f - powf(0.95f, step));
        }
        float accumulated_loss = multi_gpu_config.num_processes == 1 ? model.mean_loss : model.accumulated_mean_loss;
        printf0("step %4d/%d: train loss %f norm %.4f lr %.2e (%.2f ms, %.0f tok/s)\n",
                step + 1, train_num_batches, accumulated_loss, grad_norm, step_learning_rate,
                time_elapsed_ms, bias_corrected_ema_tokens_per_second);
        logger_log_train(&logger, step, model.mean_loss);

        // disable the profiler after 3 steps of optimization
        if (step == 3) { hipProfilerStop(); }
    }
    // add a total average, for optimizations that are only mild improvements (excluding 1st batch as warmup)
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / (train_num_batches-1) * 1000);

    // free and destroy everything
    cudaCheck(hipEventDestroy(end));
    cudaCheck(hipEventDestroy(start));
    if (run_hellaswag) { evalloader_free(&eval_loader); }
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    logger_free(&logger);
    multi_gpu_config_free(&multi_gpu_config);
    common_free(model);
    return 0;
}
#endif
